//=============================================================================
//	main.mu
//
//	The kernel manages the interface to the DLL.
//=============================================================================

//=============================================================================
//	Includes
//=============================================================================

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <sys\timeb.h>
#ifdef _TRACEAPI
#ifdef _TRACETOFILE
#include <iostream>
#include <fstream>
#endif // _TRACETOFILE
#endif // _TRACEAPI

// includes, project
#include "main.h"

#ifdef _DEBUG
#ifdef _TRACEAPI
static char s_msgbuf[256];

char* GetApiName(long lfnIdx);
#endif
#endif

//=============================================================================
//	Methods
//=============================================================================

template <class T>
long Kernel<T>::Run(long lfnIdx, T* pfInput, long lCount, T** ppfOutput, long* plCount)
{
	hipGetLastError();

#ifdef _DEBUG
#ifdef _TRACEAPI
	snprintf(s_msgbuf, 256, "%s calling CudaDnnDLL FunctionID (%ld) %s\n", (sizeof(T) == sizeof(float)) ? "FLOAT" : "DOUBLE", lfnIdx, GetApiName(lfnIdx));
	OutputDebugStringA(s_msgbuf);
#ifdef _TRACETOFILE
	std::ofstream myfile;
	myfile.open("c:\\temp\\mycaffe_debug.txt", std::ofstream::out | std::ofstream::app);
	myfile << s_msgbuf;
	myfile.close();
#endif
#endif
#endif

	if (!m_device.IsInitialized())
		return ERROR_DEVICE_NOT_INITIALIZED;

	switch (lfnIdx)
	{
		case CUDA_FN_SETDEVICE:
			return m_device.SetDevice(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SETRANDOMSEED:
			return m_device.SetRandomSeed(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_GETDEVICE:
			return m_device.GetDevice(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_RESETDEVICE:
			return m_device.ResetDevice(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SYNCHRONIZEDEVICE:
			return m_device.SynchronizeDevice(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_GETDEVICEPROP:
			return m_device.GetDeviceProperty(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_GET_REQUIRED_COMPUTE:
			return m_device.GetRequiredCompute(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CHECKMEMORYATTRIB:
			return m_device.CheckMemoryAttributes(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_GETDEVICEMEMORY:
			return m_device.GetDeviceMemory(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_DEVICE_CANACCESSPEER:
			return m_device.CanAccessPeer(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_DEVICE_ENABLEPEERACCESS:
			return m_device.EnablePeerAccess(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_DEVICE_DISABLEPEERACCESS:
			return m_device.DisablePeerAccess(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_ALLOCMEM:
			return m_device.AllocMemory(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_ALLOCMEM_HALF:
			return m_device.AllocMemoryHalf(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_FREEMEM:
			return m_device.FreeMemory(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_GETMEM:		
			return m_device.GetMemory(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SETMEM:
			return m_device.SetMemory(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SETMEMAT:
			return m_device.SetMemoryAt(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SETPIXEL:
			return m_device.SetPixel(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_COPY_DEVICE_TO_HOST:
			return m_device.CopyGpuToHostBuffer(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_COPY_HOST_TO_DEVICE:
			return m_device.CopyHostBufferToGpu(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_ALLOCHOSTBUFFER:
			return m_device.AllocHostBuffer(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_FREEHOSTBUFFER:
			return m_device.FreeHostBuffer(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_GETHOSTBUFFERCAPACITY:
			return m_device.GetHostBufferCapacity(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_GETHOSTMEM:
			return m_device.GetHostMemory(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SETHOSTMEM:
			return m_device.SetHostMemory(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CREATE_MEMORYPOINTER:
			return m_device.CreateMemoryPointer(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_FREE_MEMORYPOINTER:
			return m_device.FreeMemoryPointer(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CREATE_STREAM:
			return m_device.CreateStream(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_FREE_STREAM:
			return m_device.FreeStream(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SYNCHRONIZE_STREAM:
			return m_device.SynchronizeStream(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SYNCHRONIZE_THREAD:
			return m_device.SynchronizeThread(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CREATE_MEMTEST:
			return m_device.CreateMemoryTest(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_FREE_MEMTEST:
			return m_device.FreeMemoryTest(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_RUN_MEMTEST:
			return m_device.RunMemoryTest(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CREATE_IMAGEOP:
			return m_device.CreateImageOp(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_FREE_IMAGEOP:
			return m_device.FreeImageOp(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_DISTORTIMAGE_IMAGEOP:
			return m_device.DistortImage(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CREATE_NCCL:
			return m_device.CreateNCCL(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_FREE_NCCL:
			return m_device.FreeNCCL(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_NCCL_INIT_SINGLEPROCESS:
			return m_device.NcclInitSingleProcess(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_NCCL_INIT_MULTIPROCESS:
			return m_device.NcclInitMultiProcess(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_NCCL_BROADCAST:
			return m_device.NcclBroadcast(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_NCCL_ALLREDUCE:
			return m_device.NcclAllReduce(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_FREE_EXTENSION:
			return m_device.FreeExtension(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_CREATE_CUDNN:
			return m_device.CreateCuDNN(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_FREE_CUDNN:	
			return m_device.FreeCuDNN(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_CREATE_TENSORDESC:
			return m_device.CreateTensorDesc(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_FREE_TENSORDESC:
			return m_device.FreeTensorDesc(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_SET_TENSORDESC:
			return m_device.SetTensorDesc(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_SET_TENSORNDDESC:
			return m_device.SetTensorNdDesc(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_ADD_TENSOR:
			return m_device.AddTensor(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_CREATE_FILTERDESC:
			return m_device.CreateFilterDesc(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_FREE_FILTERDESC:
			return m_device.FreeFilterDesc(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_SET_FILTERDESC:	
			return m_device.SetFilterDesc(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_SET_FILTERNDDESC:
			return m_device.SetFilterNdDesc(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_CREATE_CONVDESC:
			return m_device.CreateConvolutionDesc(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_FREE_CONVDESC:	
			return m_device.FreeConvolutionDesc(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_SET_CONVDESC:	
			return m_device.SetConvolutionDesc(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_GET_CONVINFO:
			return m_device.GetConvolutionInfo(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_FWD_CONV:
			return m_device.ConvolutionForward(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_BWD_CONV_BIAS:
			return m_device.ConvolutionBackwardBias(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_BWD_CONV_FILTER:
			return m_device.ConvolutionBackwardFilter(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_BWD_CONV_DATA:
			return m_device.ConvolutionBackwardData(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_CREATE_POOLDESC:
			return m_device.CreatePoolingDesc(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_FREE_POOLDESC:	
			return m_device.FreePoolingDesc(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_SET_POOLDESC:	
			return m_device.SetPoolingDesc(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_POOL_FWD:	
			return m_device.PoolingForward(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_POOL_BWD:	
			return m_device.PoolingBackward(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_DERIVE_BN:
			return m_device.DeriveBatchNormDesc(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_BN_FWD:
			return m_device.BatchNormForward(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_BN_BWD:
			return m_device.BatchNormBackward(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_GET_DROPOUT_INFO:
			return m_device.GetDropoutInfo(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_CREATE_DROPOUTDESC:
			return m_device.CreateDropoutDesc(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_FREE_DROPOUTDESC:
			return m_device.FreeDropoutDesc(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_SET_DROPOUTDESC:
			return m_device.SetDropoutDesc(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_DROPOUT_FWD:
			return m_device.DropoutForward(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_DROPOUT_BWD:
			return m_device.DropoutBackward(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_CREATE_LRNDESC:
			return m_device.CreateLRNDesc(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_FREE_LRNDESC:	
			return m_device.FreeLRNDesc(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_SET_LRNDESC:	
			return m_device.SetLRNDesc(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_TANH_FWD:	
			return m_device.TanhForward(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_TANH_BWD:	
			return m_device.TanhBackward(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_ELU_FWD:
			return m_device.EluForward(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_ELU_BWD:
			return m_device.EluBackward(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_SIGMOID_FWD:	
			return m_device.SigmoidForward(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_SIGMOID_BWD:	
			return m_device.SigmoidBackward(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_RELU_FWD:	
			return m_device.ReLUForward(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_RELU_BWD:	
			return m_device.ReLUBackward(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_SOFTMAX_FWD:	
			return m_device.SoftmaxForward(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_SOFTMAX_BWD:	
			return m_device.SoftmaxBackward(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_CREATE_RNN_DATA_DESC:
			return m_device.CreateRnnDataDesc(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_FREE_RNN_DATA_DESC:
			return m_device.FreeRnnDataDesc(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_SET_RNN_DATA_DESC:
			return m_device.SetRnnDataDesc(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_CREATE_RNN_DATA_DESCEX:
			return m_device.CreateRnnDataDescEx(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_FREE_RNN_DATA_DESCEX:
			return m_device.FreeRnnDataDescEx(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_SET_RNN_DATA_DESCEX:
			return m_device.SetRnnDataDescEx(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_CREATE_RNN_DESC:
			return m_device.CreateRnnDesc(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_FREE_RNN_DESC:
			return m_device.FreeRnnDesc(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_SET_RNN_DESC:
			return m_device.SetRnnDesc(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_GET_RNN_PARAMCOUNT:
			return m_device.GetRnnParamCount(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_GET_RNN_WORKSPACECOUNT:
			return m_device.GetRnnWorkspaceCount(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_GET_RNN_LINLAYERPARAMS:
			return m_device.GetRnnLinLayerParams(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_RNN_FWD:
			return m_device.RnnForward(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_RNN_BWD_DATA:
			return m_device.RnnBackwardData(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_RNN_BWD_WTS:
			return m_device.RnnBackwardWeights(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_LRN_CC_FWD:
			return m_device.LRNForwardCC(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_LRN_CC_BWD:
			return m_device.LRNBackwardCC(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_LCN_CC_FWD:
			return m_device.LCNForwardCC(lCount, pfInput, plCount, ppfOutput);

		case CUDNN_FN_LCN_CC_BWD:
			return m_device.LCNBackwardCC(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CREATE_PCA:
			return m_device.CreatePCA(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_FREE_PCA:
			return m_device.FreePCA(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_RUN_PCA:
			return m_device.RunPCA(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CREATE_TSNE_GAUSSIAN_PERPLEXITY:			
			return m_device.CreateTsneGaussianPerplexity(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_FREE_TSNE_GAUSSIAN_PERPLEXITY:
			return m_device.FreeTsneGaussianPerplexity(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_FIND_TSNE_GAUSSIAN_PERPLEXITY:
			return m_device.FindTsneGaussianPerplexity(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CREATE_TSNE:			
			return m_device.CreateTsne(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_FREE_TSNE:
			return m_device.FreeTsne(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_TSNE_COMPUTE_GRADIENT1:
			return m_device.ComputeTsneGradient(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_TSNE_COMPUTE_ERROR1:
			return m_device.EvaluateTsneError(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SET:
			return m_device.cuda_set(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_GET:
			return m_device.cuda_get(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_COPY:
			return m_device.cuda_copy(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_COPY_SIM:
			return m_device.cuda_copy_sim(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_COPY_FILL:
			return m_device.cuda_copy_fill(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SORT:
			return m_device.cuda_sort(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_COPY_BATCH:
			return m_device.cuda_copy_batch(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_COPY_SEQUENCE:
			return m_device.cuda_copy_sequence(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_COPY_SEQUENCE2:
			return m_device.cuda_copy_sequence2(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_COPY_EXPAND:
			return m_device.cuda_copy_expand(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_GEMM:
			return m_device.cuda_gemm(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_GEMM2:
			return m_device.cuda_gemm2(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_GEMV:
			return m_device.cuda_gemv(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_GEAM:
			return m_device.cuda_geam(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_GER:
			return m_device.cuda_ger(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SET_BOUNDS:
			return m_device.cuda_set_bounds(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_AXPY:
			return m_device.cuda_axpy(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_AXPBY:
			return m_device.cuda_axpby(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MULBSX:
			return m_device.cuda_mulbsx(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_DIVBSX:
			return m_device.cuda_divbsx(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SCAL:
			return m_device.cuda_scal(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_DOT:
			return m_device.cuda_dot(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_ASUM:
			return m_device.cuda_asum(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SCALE:
			return m_device.cuda_scale(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SCALE_TO_RANGE:
			return m_device.cuda_scale_to_range(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_ERF:
			return m_device.cuda_erf(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MASK:
			return m_device.cuda_mask(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MASK_BATCH:
			return m_device.cuda_mask_batch(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_INTERP2:
			return m_device.cuda_interp2(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_ADD_SCALAR:
			return m_device.cuda_add_scalar(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_ADD:
			return m_device.cuda_add(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_ADD2:
			return m_device.cuda_add2(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SUB:
			return m_device.cuda_sub(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MUL:
			return m_device.cuda_mul(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SUB_AND_DOT:
			return m_device.cuda_sub_and_dot(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MUL_SCALAR:
			return m_device.cuda_mul_scalar(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_DIV:
			return m_device.cuda_div(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_ABS:
			return m_device.cuda_abs(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_EXP:
			return m_device.cuda_exp(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_LOG:
			return m_device.cuda_log(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_POWX:
			return m_device.cuda_powx(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SIGN:
			return m_device.cuda_sign(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SQRT:
			return m_device.cuda_sqrt(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_RECIPROCOL:
			return m_device.cuda_reciprocol(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_STUDENT:
			return m_device.cuda_student(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_LOGISTIC1:
			return m_device.cuda_logistic1(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_LOGISTIC2:
			return m_device.cuda_logistic2(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_COMPARE_SIGNS:
			return m_device.cuda_compare_signs(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_DENAN:
			return m_device.cuda_denan(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MAXVAL:
			return m_device.cuda_maxval(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MINVAL:
			return m_device.cuda_minval(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MINMAXVAL:
			return m_device.cuda_minmaxval(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MINMAXVEC:
			return m_device.cuda_minmaxvec(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_TRANSPOSE:
			return m_device.cuda_transpose(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SUMSQ:
			return m_device.cuda_sumsq(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SUMSQDIFF:
			return m_device.cuda_sumsqdiff(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SUM:
			return m_device.cuda_sum(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SQRT_SCALE:
			return m_device.cuda_sqrt_scale(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_WIDTH:
			return m_device.cuda_width(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CONTAINS_POINT:
			return m_device.cuda_contains_point(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CHANNEL_MIN:
			return m_device.cuda_channel_min(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CHANNEL_MAX:
			return m_device.cuda_channel_max(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CHANNEL_SUB:
			return m_device.cuda_channel_sub(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CHANNEL_SUM:
			return m_device.cuda_channel_sum(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CHANNEL_DIV:
			return m_device.cuda_channel_div(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CHANNEL_MUL:
			return m_device.cuda_channel_mul(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CHANNEL_MULV:
			return m_device.cuda_channel_mulv(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CHANNEL_SCALE:
			return m_device.cuda_channel_scale(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CHANNEL_DOT:
			return m_device.cuda_channel_dot(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CHANNEL_COMPARE:
			return m_device.cuda_channel_compare(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CHANNEL_FILL:
			return m_device.cuda_channel_fill(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CHANNEL_FILLFROM:
			return m_device.cuda_channel_fillfrom(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CHANNEL_COPY:
			return m_device.cuda_channel_copy(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CHANNEL_COPYALL:
			return m_device.cuda_channel_copyall(lCount, pfInput, plCount, ppfOutput);
			
		case CUDA_FN_CHANNEL_DUPLICATE:
			return m_device.cuda_channel_duplicate(lCount, pfInput, plCount, ppfOutput);
			
		case CUDA_FN_IM2COL:
			return m_device.cuda_im2col(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_IM2COL_ND:
			return m_device.cuda_im2col_nd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_COL2IM:
			return m_device.cuda_col2im(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_COL2IM_ND:
			return m_device.cuda_col2im_nd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_RNG_SETSEED:
			return m_device.cuda_rng_setseed(lCount, pfInput, plCount, ppfOutput);

		case CUDA_RNG_UNIFORM:
			return m_device.cuda_rng_uniform(lCount, pfInput, plCount, ppfOutput);

		case CUDA_RNG_GAUSSIAN:
			return m_device.cuda_rng_gaussian(lCount, pfInput, plCount, ppfOutput);

		case CUDA_RNG_BERNOULLI:
			return m_device.cuda_rng_bernoulli(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_ACCURACY_FWD:
			return m_device.cuda_accuracy_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_BATCHREIDX_FWD:
			return m_device.cuda_batchreidx_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_BATCHREIDX_BWD:
			return m_device.cuda_batchreidx_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_EMBED_FWD:
			return m_device.cuda_embed_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_EMBED_BWD:
			return m_device.cuda_embed_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_POOL_FWD:
			return m_device.cuda_pooling_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_POOL_BWD:
			return m_device.cuda_pooling_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_UNPOOL_FWD:
			return m_device.cuda_unpooling_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_UNPOOL_BWD:
			return m_device.cuda_unpooling_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CLIP_FWD:
			return m_device.cuda_clip_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CLIP_BWD:
			return m_device.cuda_clip_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_TANH_FWD:
			return m_device.cuda_tanh_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_TANH_BWD:
			return m_device.cuda_tanh_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MEAN_ERROR_LOSS_BWD:
			return m_device.cuda_mean_error_loss_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MISH_FWD:
			return m_device.cuda_mish_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MISH_BWD:
			return m_device.cuda_mish_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_GELU_FWD:
			return m_device.cuda_gelu_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_GELU_BWD:
			return m_device.cuda_gelu_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SIGMOID_FWD:
			return m_device.cuda_sigmoid_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SIGMOID_BWD:
			return m_device.cuda_sigmoid_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SWISH_BWD:
			return m_device.cuda_swish_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_RELU_FWD:
			return m_device.cuda_relu_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_RELU_BWD:
			return m_device.cuda_relu_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_ELU_FWD:
			return m_device.cuda_elu_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_ELU_BWD:
			return m_device.cuda_elu_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_DROPOUT_FWD:
			return m_device.cuda_dropout_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_DROPOUT_BWD:
			return m_device.cuda_dropout_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_BNLL_FWD:
			return m_device.cuda_bnll_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_BNLL_BWD:
			return m_device.cuda_bnll_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_PRELU_FWD:
			return m_device.cuda_prelu_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_PRELU_BWD:
			return m_device.cuda_prelu_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_PRELU_BWD_PARAM:
			return m_device.cuda_prelu_bwd_param(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_NLLLOSS_FWD:
			return m_device.cuda_nllloss_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_NLLLOSS_BWD:
			return m_device.cuda_nllloss_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SOFTMAXLOSS_FWD:
			return m_device.cuda_softmaxloss_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SOFTMAXLOSS_BWD:
			return m_device.cuda_softmaxloss_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MIN_FWD:
			return m_device.cuda_min_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MIN_BWD:
			return m_device.cuda_min_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MAX_FWD:
			return m_device.cuda_max_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MAX_BWD:
			return m_device.cuda_max_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CROP_FWD:
			return m_device.cuda_crop_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CROP_BWD:
			return m_device.cuda_crop_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CONCAT_FWD:
			return m_device.cuda_concat_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CONCAT_BWD:
			return m_device.cuda_concat_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SLICE_FWD:
			return m_device.cuda_slice_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SLICE_BWD:
			return m_device.cuda_slice_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_TILE_FWD:
			return m_device.cuda_tile_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_TILE_BWD:
			return m_device.cuda_tile_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_BIAS_FWD:
			return m_device.cuda_bias_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SCALE_FWD:
			return m_device.cuda_scale_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_THRESHOLD_FWD:
			return m_device.cuda_threshold_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CLL_BWD:
			return m_device.cuda_cll_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SMOOTHL1_FWD:
			return m_device.cuda_smoothl1_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SMOOTHL1_BWD:
			return m_device.cuda_smoothl1_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SERF_FWD:
			return m_device.cuda_serf_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SERF_BWD:
			return m_device.cuda_serf_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_PERMUTE:
			return m_device.cuda_permute(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_GATHER_FWD:
			return m_device.cuda_gather_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_GATHER_BWD:
			return m_device.cuda_gather_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_LRN_FILLSCALE:
			return m_device.cuda_lrn_fillscale(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_LRN_COMPUTEOUTPUT:
			return m_device.cuda_lrn_computeoutput(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_LRN_COMPUTEDIFF:
			return m_device.cuda_lrn_computediff(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_LSTM_FWD:
			return m_device.cuda_lstm_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_LSTM_BWD:
			return m_device.cuda_lstm_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_LSTM_UNIT_FWD:
			return m_device.cuda_lstm_unit_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_LSTM_UNIT_BWD:
			return m_device.cuda_lstm_unit_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MATH_FWD:
			return m_device.cuda_math_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MATH_BWD:
			return m_device.cuda_math_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_COEFF_SUM_FWD:
			return m_device.cuda_coeff_sum_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_COEFF_SUM_BWD:
			return m_device.cuda_coeff_sum_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_COEFF_SUB_FWD:
			return m_device.cuda_coeff_sub_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_COEFF_SUB_BWD:
			return m_device.cuda_coeff_sub_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SIGMOID_CROSS_ENTROPY_FWD:
			return m_device.cuda_sigmoid_cross_entropy_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SIGMOID_CROSS_ENTROPY_BWD:
			return m_device.cuda_sigmoid_cross_entropy_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SOFTMAX_CROSS_ENTROPY_FWD:
			return m_device.cuda_softmax_cross_entropy_fwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SOFTMAX_CROSS_ENTROPY_BWD:
			return m_device.cuda_softmax_cross_entropy_bwd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SGD_UPDATE:
			return m_device.cuda_sgd_update(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_NESTEROV_UPDATE:
			return m_device.cuda_nesterov_update(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_ADAGRAD_UPDATE:
			return m_device.cuda_adagrad_update(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_ADADELTA_UPDATE:
			return m_device.cuda_adadelta_update(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_ADAM_UPDATE:
			return m_device.cuda_adam_update(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_ADAMW_UPDATE:
			return m_device.cuda_adamw_update(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_RMSPROP_UPDATE:
			return m_device.cuda_rmsprop_update(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_COMBINE_DATA:
			return m_device.cuda_combine_data(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MTX_SET_DIAGONAL:
			return m_device.cuda_mtx_set_diagonal(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MTX_SET_DIAGONAL2:
			return m_device.cuda_mtx_set_diagonal2(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MTX_ADD_VECTOR:
			return m_device.cuda_mtx_add_vector(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MTX_TRANSPOSE_OP:
			return m_device.cuda_mtx_transpose_op(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MTX_AGGREGATE_COLS:
			return m_device.cuda_mtx_aggregate_cols(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MTX_AGGREGATE_ROWS:
			return m_device.cuda_mtx_aggregate_rows(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MTX_TRANSPOSE:
			return m_device.cuda_mtx_transpose(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MTX_MEANCENTER_BY_COL:
			return m_device.cuda_mtx_meancenter_by_column(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MTX_EUCLIDEAN_DIST:
			return m_device.cuda_mtx_euclidean_dist(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MTX_DOT:
			return m_device.cuda_mtx_dot(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MTX_MEAN:
			return m_device.cuda_mtx_mean(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MTX_STDEV:
			return m_device.cuda_mtx_stdev(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_MTX_CORRELATION:
			return m_device.cuda_mtx_correlation(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_TSNE_UPDATE:
			return m_device.cuda_tsne_update(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_TSNE_UPDATE_GRAD:
			return m_device.cuda_tsne_update_grad(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_TSNE_COMPUTE_EXACT_ERROR:
			return m_device.cuda_tsne_compute_exact_error(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_TSNE_COMPUTE_SQUARED_EUCLIDEAN_DISTANCE:
			return m_device.cuda_tsne_compute_squared_euclidean_distance(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_TSNE_COMPUTE_Q_MATRIX:
			return m_device.cuda_tsne_compute_q_matrix(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_TSNE_COMPUTE_EXACT_GRADIENT:
			return m_device.cuda_tsne_compute_exact_gradient(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_TSNE_SYMMETRIZE_MATRIX:
			return m_device.cuda_tsne_symmetrize_matrix(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_TSNE_COMPUTE_KNN_BOUNDS:
			return m_device.cuda_tsne_compute_knn_bounds(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_GUASSIAN_BLUR:
			return m_device.cuda_guassian_blur(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CALC_DFT:
			return m_device.cuda_calc_dft(lCount, pfInput, plCount, ppfOutput);
	
		case CUDA_FN_HAMMING_DIFF:
			return m_device.cuda_hamming_diff(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CALC_BATCH_DIST:
			return m_device.cuda_calc_batch_dist(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CREATE_SSD:
			return m_device.CreateSsd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_FREE_SSD:
			return m_device.FreeSsd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SETUP_SSD:
			return m_device.SetupSsd(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SSD_FWD_MULTIBOXLOSS:
			return m_device.SsdMultiboxLossForward(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SSD_ENCODE_LOCPRED:
			return m_device.SsdEncodeLocPrediction(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_SSD_ENCODE_CONFPRED:
			return m_device.SsdEncodeConfPrediction(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_CREATE_LAYERNORM:
			return m_device.CreateLayerNorm(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_FREE_LAYERNORM:
			return m_device.FreeLayerNorm(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_LAYERNORM_FWD:
			return m_device.LayerNormForward(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_LAYERNORM_BWD:
			return m_device.LayerNormBackward(lCount, pfInput, plCount, ppfOutput);

		case CUDA_FN_DEBUG:
			return 0;

		default:
			return ERROR_PARAM_OUT_OF_RANGE;
	}
}

template long Kernel<double>::Run(long lfnIdx, double* pfInput, long lCount, double** ppfOutput, long* plCount);
template long Kernel<float>::Run(long lfnIdx, float* pfInput, long lCount, float** ppfOutput, long* plCount);


template <class T>
long Kernel<T>::Run(long lfnIdx, T* pfInput, long lCount, LPTSTR pszInput, T** ppfOutput, long* plCount)
{
	hipGetLastError();

#ifdef _DEBUG
#ifdef _TRACEAPI
	snprintf(s_msgbuf, 256, "%s calling CudaDnnDLL FunctionID (%ld) %s\n", (sizeof(T) == sizeof(float)) ? "FLOAT" : "DOUBLE", lfnIdx, GetApiName(lfnIdx));
	OutputDebugStringA(s_msgbuf);
#ifdef _TRACETOFILE
	std::ofstream myfile;
	myfile.open("c:\\temp\\mycaffe_debug.txt", std::ofstream::out | std::ofstream::app);
	myfile << s_msgbuf;
	myfile.close();
#endif
#endif
#endif

	//switch (lfnIdx)
	//{
	//	default:
			return ERROR_NOT_SUPPORTED;
	//}
}

template long Kernel<double>::Run(long lfnIdx, double* pfInput, long lCount, LPTSTR pszInput, double** ppfOutput, long* plCount);
template long Kernel<float>::Run(long lfnIdx, float* pfInput, long lCount, LPTSTR pszInput, float** ppfOutput, long* plCount);


template <class T>
long Kernel<T>::Run(long lfnIdx, T* pfInput, long lCount, T** ppfOutput, long* plCount, LPTSTR szErr, long lMaxErr)
{
	hipGetLastError();

#ifdef _DEBUG
#ifdef _TRACEAPI
	snprintf(s_msgbuf, 256, "%s calling CudaDnnDLL FunctionID (%ld) %s\n", (sizeof(T) == sizeof(float)) ? "FLOAT" : "DOUBLE", lfnIdx, GetApiName(lfnIdx));
	OutputDebugStringA(s_msgbuf);
#ifdef _TRACETOFILE
	std::ofstream myfile;
	myfile.open("c:\\temp\\mycaffe_debug.txt", std::ofstream::out | std::ofstream::app);
	myfile << s_msgbuf;
	myfile.close();
#endif
#endif
#endif

	switch (lfnIdx)
	{
		case CUDA_FN_EXTENSION_RUN:
			return m_device.ExtensionRun(lCount, pfInput, plCount, ppfOutput, szErr, lMaxErr);

		default:
			return ERROR_PARAM_OUT_OF_RANGE;
	}
}

template long Kernel<double>::Run(long lfnIdx, double* pfInput, long lCount, double** ppfOutput, long* plCount, LPTSTR szErr, long lMaxErr);
template long Kernel<float>::Run(long lfnIdx, float* pfInput, long lCount, float** ppfOutput, long* plCount, LPTSTR szErr, long lMaxErr);


#ifdef _DEBUG
#ifdef _TRACEAPI
char* GetApiName(long lfnIdx)
{
	switch (lfnIdx)
	{
	case CUDA_FN_SETDEVICE:
		return "CUDA_FN_SETDEVICE";

	case CUDA_FN_SETRANDOMSEED:
		return "CUDA_FN_SETRANDOMSEED";

	case CUDA_FN_GETDEVICE:
		return "CUDA_FN_GETDEVICE";

	case CUDA_FN_RESETDEVICE:
		return "CUDA_FN_RESETDEVICE";

	case CUDA_FN_SYNCHRONIZEDEVICE:
		return "CUDA_FN_SYNCHRONIZEDEVICE";

	case CUDA_FN_GETDEVICEPROP:
		return "CUDA_FN_GETDEVICEPROP";

	case CUDA_FN_GET_REQUIRED_COMPUTE:
		return "CUDA_FN_GET_REQUIRED_COMPUTE";

	case CUDA_FN_CHECKMEMORYATTRIB:
		return "CUDA_FN_CHECKMEMORYATTRIB";

	case CUDA_FN_GETDEVICEMEMORY:
		return "CUDA_FN_GETDEVICEMEMORY";

	case CUDA_FN_DEVICE_CANACCESSPEER:
		return "CUDA_FN_DEVICE_CANACCESSPEER";

	case CUDA_FN_DEVICE_ENABLEPEERACCESS:
		return "CUDA_FN_DEVICE_ENABLEPEERACCESS";

	case CUDA_FN_DEVICE_DISABLEPEERACCESS:
		return "CUDA_FN_DEVICE_DISABLEPEERACCESS";

	case CUDA_FN_ALLOCMEM:
		return "CUDA_FN_ALLOCMEM";

	case CUDA_FN_FREEMEM:
		return "CUDA_FN_FREEMEM";

	case CUDA_FN_GETMEM:
		return "CUDA_FN_GETMEM";

	case CUDA_FN_SETMEM:
		return "CUDA_FN_SETMEM";

	case CUDA_FN_SETMEMAT:
		return "CUDA_FN_SETMEMAT";

	case CUDA_FN_SETPIXEL:
		return "CUDA_FN_SETPIXEL";

	case CUDA_FN_COPY_DEVICE_TO_HOST:
		return "CUDA_FN_COPY_DEVICE_TO_HOST";

	case CUDA_FN_COPY_HOST_TO_DEVICE:
		return "CUDA_FN_COPY_HOST_TO_DEVICE";

	case CUDA_FN_ALLOCHOSTBUFFER:
		return "CUDA_FN_ALLOCHOSTBUFFER";

	case CUDA_FN_FREEHOSTBUFFER:
		return "CUDA_FN_FREEHOSTBUFFER";

	case CUDA_FN_GETHOSTMEM:
		return "CUDA_FN_GETHOSTMEM";

	case CUDA_FN_SETHOSTMEM:
		return "CUDA_FN_SETHOSTMEM";

	case CUDA_FN_CREATE_MEMORYPOINTER:
		return "CUDA_FN_CREATE_MEMORYPOINTER";

	case CUDA_FN_FREE_MEMORYPOINTER:
		return "CUDA_FN_FREE_MEMORYPOINTER";

	case CUDA_FN_CREATE_STREAM:
		return "CUDA_FN_CREATE_STREAM";

	case CUDA_FN_FREE_STREAM:
		return "CUDA_FN_FREE_STREAM";

	case CUDA_FN_SYNCHRONIZE_STREAM:
		return "CUDA_FN_SYNCHRONIZE_STREAM";

	case CUDA_FN_SYNCHRONIZE_THREAD:
		return "CUDA_FN_SYNCHRONIZE_THREAD";

	case CUDA_FN_CREATE_MEMTEST:
		return "CUDA_FN_CREATE_MEMTEST";

	case CUDA_FN_FREE_MEMTEST:
		return "CUDA_FN_FREE_MEMTEST";

	case CUDA_FN_RUN_MEMTEST:
		return "CUDA_FN_RUN_MEMTEST";

	case CUDA_FN_CREATE_IMAGEOP:
		return "CUDA_FN_CREATE_IMAGEOP";

	case CUDA_FN_FREE_IMAGEOP:
		return "CUDA_FN_FREE_IMAGEOP";

	case CUDA_FN_DISTORTIMAGE_IMAGEOP:
		return "CUDA_FN_DISTORTIMAGE_IMAGEOP";

	case CUDA_FN_CREATE_NCCL:
		return "CUDA_FN_CREATE_NCCL";

	case CUDA_FN_FREE_NCCL:
		return "CUDA_FN_FREE_NCCL";

	case CUDA_FN_NCCL_INIT_SINGLEPROCESS:
		return "CUDA_FN_NCCL_INIT_SINGLEPROCESS";

	case CUDA_FN_NCCL_INIT_MULTIPROCESS:
		return "CUDA_FN_NCCL_INIT_MULTIPROCESS";

	case CUDA_FN_NCCL_BROADCAST:
		return "CUDA_FN_NCCL_BROADCAST";

	case CUDA_FN_NCCL_ALLREDUCE:
		return "CUDA_FN_NCCL_ALLREDUCE";

	case CUDA_FN_CREATE_EXTENSION:
		return "CUDA_FN_CREATE_EXTENSION";

	case CUDA_FN_FREE_EXTENSION:
		return "CUDA_FN_FREE_EXTENSION";

	case CUDA_FN_EXTENSION_RUN:
		return "CUDA_FN_EXTENSION_RUN";

	case CUDNN_FN_CREATE_CUDNN:
		return "CUDNN_FN_CREATE_CUDNN";

	case CUDNN_FN_FREE_CUDNN:
		return "CUDNN_FN_FREE_CUDNN";

	case CUDNN_FN_CREATE_TENSORDESC:
		return "CUDNN_FN_CREATE_TENSORDESC";

	case CUDNN_FN_FREE_TENSORDESC:
		return "CUDNN_FN_FREE_TENSORDESC";

	case CUDNN_FN_SET_TENSORDESC:
		return "CUDNN_FN_SET_TENSORDESC";

	case CUDNN_FN_SET_TENSORNDDESC:
		return "CUDNN_FN_SET_TENSORNDDESC";

	case CUDNN_FN_ADD_TENSOR:
		return "CUDNN_FN_ADD_TENSOR";

	case CUDNN_FN_CREATE_FILTERDESC:
		return "CUDNN_FN_CREATE_FILTERDESC";

	case CUDNN_FN_FREE_FILTERDESC:
		return "CUDNN_FN_FREE_FILTERDESC";

	case CUDNN_FN_SET_FILTERDESC:
		return "CUDNN_FN_SET_FILTERDESC";

	case CUDNN_FN_SET_FILTERNDDESC:
		return "CUDNN_FN_SET_FILTERNDDESC";

	case CUDNN_FN_CREATE_CONVDESC:
		return "CUDNN_FN_CREATE_CONVDESC";

	case CUDNN_FN_FREE_CONVDESC:
		return "CUDNN_FN_FREE_CONVDESC";

	case CUDNN_FN_SET_CONVDESC:
		return "CUDNN_FN_SET_CONVDESC";

	case CUDNN_FN_GET_CONVINFO:
		return "CUDNN_FN_GET_CONVINFO";

	case CUDNN_FN_FWD_CONV:
		return "CUDNN_FN_FWD_CONV";

	case CUDNN_FN_BWD_CONV_BIAS:
		return "CUDNN_FN_BWD_CONV_BIAS";

	case CUDNN_FN_BWD_CONV_FILTER:
		return "CUDNN_FN_BWD_CONV_FILTER";

	case CUDNN_FN_BWD_CONV_DATA:
		return "CUDNN_FN_BWD_CONV_DATA";

	case CUDNN_FN_CREATE_POOLDESC:
		return "CUDNN_FN_CREATE_POOLDESC";

	case CUDNN_FN_FREE_POOLDESC:
		return "CUDNN_FN_FREE_POOLDESC";

	case CUDNN_FN_SET_POOLDESC:
		return "CUDNN_FN_SET_POOLDESC";

	case CUDNN_FN_POOL_FWD:
		return "CUDNN_FN_POOL_FWD";

	case CUDNN_FN_POOL_BWD:
		return "CUDNN_FN_POOL_BWD";

	case CUDNN_FN_GET_DROPOUT_INFO:
		return "CUDNN_FN_GET_DROPOUT_INFO";

	case CUDNN_FN_CREATE_DROPOUTDESC:
		return "CUDNN_FN_CREATE_DROPOUTDESC";

	case CUDNN_FN_FREE_DROPOUTDESC:
		return "CUDNN_FN_FREE_DROPOUTDESC";

	case CUDNN_FN_SET_DROPOUTDESC:
		return "CUDNN_FN_SET_DROPOUTDESC";

	case CUDNN_FN_DROPOUT_FWD:
		return "CUDNN_FN_DROPOUT_FWD";

	case CUDNN_FN_DROPOUT_BWD:
		return "CUDNN_FN_DROPOUT_BWD";

	case CUDNN_FN_CREATE_LRNDESC:
		return "CUDNN_FN_CREATE_LRNDESC";

	case CUDNN_FN_FREE_LRNDESC:
		return "CUDNN_FN_FREE_LRNDESC";

	case CUDNN_FN_SET_LRNDESC:
		return "CUDNN_FN_SET_LRNDESC";

	case CUDNN_FN_TANH_FWD:
		return "CUDNN_FN_TANH_FWD";

	case CUDNN_FN_TANH_BWD:
		return "CUDNN_FN_TANH_BWD";

	case CUDNN_FN_SIGMOID_FWD:
		return "CUDNN_FN_SIGMOID_FWD";

	case CUDNN_FN_SIGMOID_BWD:
		return "CUDNN_FN_SIGMOID_BWD";

	case CUDNN_FN_RELU_FWD:
		return "CUDNN_FN_RELU_FWD";

	case CUDNN_FN_RELU_BWD:
		return "CUDNN_FN_RELU_BWD";

	case CUDNN_FN_SOFTMAX_FWD:
		return "CUDNN_FN_SOFTMAX_FWD";

	case CUDNN_FN_SOFTMAX_BWD:
		return "CUDNN_FN_SOFTMAX_BWD";

	case CUDNN_FN_CREATE_RNN_DATA_DESC:
		return "CUDNN_FN_CREATE_RNN_DATA_DESC";

	case CUDNN_FN_FREE_RNN_DATA_DESC:
		return "CUDNN_FN_FREE_RNN_DATA_DESC";

	case CUDNN_FN_SET_RNN_DATA_DESC:
		return "CUDNN_FN_SET_RNN_DATA_DESC";

	case CUDNN_FN_CREATE_RNN_DATA_DESCEX:
		return "CUDNN_FN_CREATE_RNN_DATA_DESCEX";

	case CUDNN_FN_FREE_RNN_DATA_DESCEX:
		return "CUDNN_FN_FREE_RNN_DATA_DESCEX";

	case CUDNN_FN_SET_RNN_DATA_DESCEX:
		return "CUDNN_FN_SET_RNN_DATA_DESCEX";

	case CUDNN_FN_CREATE_RNN_DESC:
		return "CUDNN_FN_CREATE_RNN_DESC";

	case CUDNN_FN_FREE_RNN_DESC:
		return "CUDNN_FN_FREE_RNN_DESC";

	case CUDNN_FN_SET_RNN_DESC:
		return "CUDNN_FN_SET_RNN_DESC";

	case CUDNN_FN_GET_RNN_PARAMCOUNT:
		return "CUDNN_FN_GET_RNN_PARAMCOUNT";

	case CUDNN_FN_GET_RNN_WORKSPACECOUNT:
		return "CUDNN_FN_GET_RNN_WORKSPACECOUNT";

	case CUDNN_FN_GET_RNN_LINLAYERPARAMS:
		return "CUDNN_FN_GET_RNN_LINLAYERPARAMS";

	case CUDNN_FN_RNN_FWD:
		return "CUDNN_FN_RNN_FWD";

	case CUDNN_FN_RNN_BWD_DATA:
		return "CUDNN_FN_RNN_BWD_DATA";

	case CUDNN_FN_RNN_BWD_WTS:
		return "CUDNN_FN_RNN_BWD_WTS";

	case CUDNN_FN_LRN_CC_FWD:
		return "CUDNN_FN_LRN_CC_FWD";

	case CUDNN_FN_LRN_CC_BWD:
		return "CUDNN_FN_LRN_CC_BWD";

	case CUDNN_FN_LCN_CC_FWD:
		return "CUDNN_FN_LCN_CC_FWD";

	case CUDNN_FN_LCN_CC_BWD:
		return "CUDNN_FN_LCN_CC_BWD";

	case CUDA_FN_CREATE_PCA:
		return "CUDA_FN_CREATE_PCA";

	case CUDA_FN_FREE_PCA:
		return "CUDA_FN_FREE_PCA";

	case CUDA_FN_RUN_PCA:
		return "CUDA_FN_RUN_PCA";

	case CUDA_FN_CREATE_TSNE_GAUSSIAN_PERPLEXITY:
		return "CUDA_FN_CREATE_TSNE_GAUSSIAN_PERPLEXITY";

	case CUDA_FN_FREE_TSNE_GAUSSIAN_PERPLEXITY:
		return "CUDA_FN_FREE_TSNE_GAUSSIAN_PERPLEXITY";

	case CUDA_FN_FIND_TSNE_GAUSSIAN_PERPLEXITY:
		return "CUDA_FN_FIND_TSNE_GAUSSIAN_PERPLEXITY";

	case CUDA_FN_CREATE_TSNE:
		return "CUDA_FN_CREATE_TSNE";

	case CUDA_FN_FREE_TSNE:
		return "CUDA_FN_FREE_TSNE";

	case CUDA_FN_TSNE_COMPUTE_GRADIENT1:
		return "CUDA_FN_TSNE_COMPUTE_GRADIENT1";

	case CUDA_FN_TSNE_COMPUTE_ERROR1:
		return "CUDA_FN_TSNE_COMPUTE_ERROR1";

	case CUDA_FN_SET:
		return "CUDA_FN_SET";

	case CUDA_FN_GET:
		return "CUDA_FN_GET";

	case CUDA_FN_COPY:
		return "CUDA_FN_COPY";

	case CUDA_FN_COPY_SIM:
		return "CUDA_FN_COPY_SIM";

	case CUDA_FN_COPY_FILL:
		return "CUDA_FN_COPY_FILL";

	case CUDA_FN_SORT:
		return "CUDA_FN_SORT";

	case CUDA_FN_COPY_BATCH:
		return "CUDA_FN_COPY_BATCH";

	case CUDA_FN_COPY_SEQUENCE:
		return "CUDA_FN_COPY_SEQUENCE";

	case CUDA_FN_COPY_SEQUENCE2:
		return "CUDA_FN_COPY_SEQUENCE2";

	case CUDA_FN_COPY_EXPAND:
		return "CUDA_FN_COPY_EXPAND";

	case CUDA_FN_GEMM:
		return "CUDA_FN_GEMM";

	case CUDA_FN_GEMM2:
		return "CUDA_FN_GEMM2";

	case CUDA_FN_GEMV:
		return "CUDA_FN_GEMV";

	case CUDA_FN_GEAM:
		return "CUDA_FN_GEAM";

	case CUDA_FN_GER:
		return "CUDA_FN_GER";

	case CUDA_FN_SET_BOUNDS:
		return "CUDA_FN_SET_BOUNDS";

	case CUDA_FN_AXPY:
		return "CUDA_FN_AXPY";

	case CUDA_FN_AXPBY:
		return "CUDA_FN_AXPBY";

	case CUDA_FN_MULBSX:
		return "CUDA_FN_MULBSX";

	case CUDA_FN_DIVBSX:
		return "CUDA_FN_DIVBSX";

	case CUDA_FN_SCAL:
		return "CUDA_FN_SCAL";

	case CUDA_FN_DOT:
		return "CUDA_FN_DOT";

	case CUDA_FN_ASUM:
		return "CUDA_FN_ASUM";

	case CUDA_FN_SCALE:
		return "CUDA_FN_SCALE";

	case CUDA_FN_SCALE_TO_RANGE:
		return "CUDA_FN_SCALE_TO_RANGE";

	case CUDA_FN_ERF:
		return "CUDA_FN_ERF";

	case CUDA_FN_MASK:
		return "CUDA_FN_MASK";

	case CUDA_FN_MASK_BATCH:
		return "CUDA_FN_MASK_BATCH";

	case CUDA_FN_INTERP2:
		return "CUDA_FN_INTERP2";

	case CUDA_FN_ADD_SCALAR:
		return "CUDA_FN_ADD_SCALAR";

	case CUDA_FN_ADD:
		return "CUDA_FN_ADD";

	case CUDA_FN_ADD2:
		return "CUDA_FN_ADD2";

	case CUDA_FN_SUB:
		return "CUDA_FN_SUB";

	case CUDA_FN_MUL:
		return "CUDA_FN_MUL";

	case CUDA_FN_SUB_AND_DOT:
		return "CUDA_FN_SUB_AND_DOT";

	case CUDA_FN_MUL_SCALAR:
		return "CUDA_FN_MUL_SCALAR";

	case CUDA_FN_DIV:
		return "CUDA_FN_DIV";

	case CUDA_FN_ABS:
		return "CUDA_FN_ABS";

	case CUDA_FN_EXP:
		return "CUDA_FN_EXP";

	case CUDA_FN_LOG:
		return "CUDA_FN_LOG";

	case CUDA_FN_POWX:
		return "CUDA_FN_POWX";

	case CUDA_FN_SIGN:
		return "CUDA_FN_SIGN";

	case CUDA_FN_SQRT:
		return "CUDA_FN_SQRT";

	case CUDA_FN_SQRT_SCALE:
		return "CUDA_FN_SQRT_SCALE";

	case CUDA_FN_RECIPROCOL:
		return "CUDA_FN_RECIPROCOL";

	case CUDA_FN_STUDENT:
		return "CUDA_FN_STUDENT";

	case CUDA_FN_LOGISTIC1:
		return "CUDA_FN_LOGISTIC1";

	case CUDA_FN_LOGISTIC2:
		return "CUDA_FN_LOGISTIC2";

	case CUDA_FN_COMPARE_SIGNS:
		return "CUDA_FN_COMPARE_SIGNS";

	case CUDA_FN_DENAN:
		return "CUDA_FN_DENAN";

	case CUDA_FN_MAXVAL:
		return "CUDA_FN_MAXVAL";

	case CUDA_FN_MINVAL:
		return "CUDA_FN_MINVAL";

	case CUDA_FN_MINMAXVAL:
		return "CUDA_FN_MINMAXVAL";

	case CUDA_FN_MINMAXVEC:
		return "CUDA_FN_MINMAXVEC";

	case CUDA_FN_TRANSPOSE:
		return "CUDA_FN_TRANSPOSE";

	case CUDA_FN_SUMSQ:
		return "CUDA_FN_SUMSQ";

	case CUDA_FN_SUMSQDIFF:
		return "CUDA_FN_SUMSQDIFF";

	case CUDA_FN_WIDTH:
		return "CUDA_FN_WIDTH";

	case CUDA_FN_CONTAINS_POINT:
		return "CUDA_FN_CONTAINS_POINT";

	case CUDA_FN_CHANNEL_MIN:
		return "CUDA_FN_CHANNEL_MIN";

	case CUDA_FN_CHANNEL_MAX:
		return "CUDA_FN_CHANNEL_MAX";

	case CUDA_FN_CHANNEL_SUB:
		return "CUDA_FN_CHANNEL_SUB";

	case CUDA_FN_CHANNEL_SUM:
		return "CUDA_FN_CHANNEL_SUM";

	case CUDA_FN_CHANNEL_DIV:
		return "CUDA_FN_CHANNEL_DIV";

	case CUDA_FN_CHANNEL_MUL:
		return "CUDA_FN_CHANNEL_MUL";

	case CUDA_FN_CHANNEL_MULV:
		return "CUDA_FN_CHANNEL_MULV";

	case CUDA_FN_CHANNEL_SCALE:
		return "CUDA_FN_CHANNEL_SCALE";

	case CUDA_FN_CHANNEL_DOT:
		return "CUDA_FN_CHANNEL_DOT";

	case CUDA_FN_CHANNEL_COMPARE:
		return "CUDA_FN_CHANNEL_COMPARE";

	case CUDA_FN_CHANNEL_FILL:
		return "CUDA_FN_CHANNEL_FILL";

	case CUDA_FN_CHANNEL_FILLFROM:
		return "CUDA_FN_CHANNEL_FILLFROM";

	case CUDA_FN_CHANNEL_COPY:
		return "CUDA_FN_CHANNEL_COPY";

	case CUDA_FN_CHANNEL_COPYALL:
		return "CUDA_FN_CHANNEL_COPYALL";

	case CUDA_FN_CHANNEL_DUPLICATE:
		return "CUDA_FN_CHANNEL_DUPLICATE";

	case CUDA_FN_IM2COL:
		return "CUDA_FN_IM2COL";

	case CUDA_FN_IM2COL_ND:
		return "CUDA_FN_IM2COL_ND";

	case CUDA_FN_COL2IM:
		return "CUDA_FN_COL2IM";

	case CUDA_FN_COL2IM_ND:
		return "CUDA_FN_COL2IM_ND";

	case CUDA_RNG_SETSEED:
		return "CUDA_RNG_SETSEED";

	case CUDA_RNG_UNIFORM:
		return "CUDA_RNG_UNIFORM";

	case CUDA_RNG_GAUSSIAN:
		return "CUDA_RNG_GAUSSIAN";

	case CUDA_RNG_BERNOULLI:
		return "CUDA_RNG_BERNOULLI";

	case CUDA_FN_ACCURACY_FWD:
		return "CUDA_FN_ACCURACY_FWD";

	case CUDA_FN_BATCHREIDX_FWD:
		return "CUDA_FN_BATCHREIDX_FWD";

	case CUDA_FN_BATCHREIDX_BWD:
		return "CUDA_FN_BATCHREIDX_BWD";

	case CUDA_FN_EMBED_FWD:
		return "CUDA_FN_EMBED_FWD";

	case CUDA_FN_EMBED_BWD:
		return "CUDA_FN_EMBED_BWD";

	case CUDA_FN_POOL_FWD:
		return "CUDA_FN_POOL_FWD";

	case CUDA_FN_POOL_BWD:
		return "CUDA_FN_POOL_BWD";

	case CUDA_FN_UNPOOL_FWD:
		return "CUDA_FN_UNPOOL_FWD";

	case CUDA_FN_UNPOOL_BWD:
		return "CUDA_FN_UNPOOL_BWD";

	case CUDA_FN_TANH_FWD:
		return "CUDA_FN_TANH_FWD";

	case CUDA_FN_TANH_BWD:
		return "CUDA_FN_TANH_BWD";

	case CUDA_FN_MEAN_ERROR_LOSS_BWD:
		return "CUDA_FN_MEAN_ERROR_LOSS_BWD";

	case CUDA_FN_MISH_FWD:
		return "CUDA_FN_MISH_FWD";

	case CUDA_FN_MISH_BWD:
		return "CUDA_FN_MISH_BWD";

	case CUDA_FN_GELU_FWD:
		return "CUDA_FN_GELU_FWD";

	case CUDA_FN_GELU_BWD:
		return "CUDA_FN_GELU_BWD";

	case CUDA_FN_SIGMOID_FWD:
		return "CUDA_FN_SIGMOID_FWD";

	case CUDA_FN_SIGMOID_BWD:
		return "CUDA_FN_SIGMOID_BWD";

	case CUDA_FN_SWISH_BWD:
		return "CUDA_FN_SWISH_BWD";

	case CUDA_FN_RELU_FWD:
		return "CUDA_FN_RELU_FWD";

	case CUDA_FN_RELU_BWD:
		return "CUDA_FN_RELU_BWD";

	case CUDA_FN_ELU_FWD:
		return "CUDA_FN_ELU_FWD";

	case CUDA_FN_ELU_BWD:
		return "CUDA_FN_ELU_BWD";

	case CUDA_FN_DROPOUT_FWD:
		return "CUDA_FN_DROPOUT_FWD";

	case CUDA_FN_DROPOUT_BWD:
		return "CUDA_FN_DROPOUT_BWD";

	case CUDA_FN_BNLL_FWD:
		return "CUDA_FN_BNLL_FWD";

	case CUDA_FN_BNLL_BWD:
		return "CUDA_FN_BNLL_BWD";

	case CUDA_FN_PRELU_FWD:
		return "CUDA_FN_PRELU_FWD";

	case CUDA_FN_PRELU_BWD:
		return "CUDA_FN_PRELU_BWD";

	case CUDA_FN_PRELU_BWD_PARAM:
		return "CUDA_FN_PRELU_BWD_PARAM";

	case CUDA_FN_NLLLOSS_FWD:
		return "CUDA_FN_NLLLOSS_FWD";

	case CUDA_FN_NLLLOSS_BWD:
		return "CUDA_FN_NLLLOSS_BWD";

	case CUDA_FN_SOFTMAXLOSS_FWD:
		return "CUDA_FN_SOFTMAXLOSS_FWD";

	case CUDA_FN_SOFTMAXLOSS_BWD:
		return "CUDA_FN_SOFTMAXLOSS_BWD";

	case CUDA_FN_MIN_FWD:
		return "CUDA_FN_MIN_FWD";

	case CUDA_FN_MIN_BWD:
		return "CUDA_FN_MIN_BWD";

	case CUDA_FN_MAX_FWD:
		return "CUDA_FN_MAX_FWD";

	case CUDA_FN_MAX_BWD:
		return "CUDA_FN_MAX_BWD";

	case CUDA_FN_CROP_FWD:
		return "CUDA_FN_CROP_FWD";

	case CUDA_FN_CROP_BWD:
		return "CUDA_FN_CROP_BWD";

	case CUDA_FN_CONCAT_FWD:
		return "CUDA_FN_CONCAT_FWD";

	case CUDA_FN_CONCAT_BWD:
		return "CUDA_FN_CONCAT_BWD";

	case CUDA_FN_SLICE_FWD:
		return "CUDA_FN_SLICE_FWD";

	case CUDA_FN_SLICE_BWD:
		return "CUDA_FN_SLICE_BWD";

	case CUDA_FN_TILE_FWD:
		return "CUDA_FN_TILE_FWD";

	case CUDA_FN_TILE_BWD:
		return "CUDA_FN_TILE_BWD";

	case CUDA_FN_BIAS_FWD:
		return "CUDA_FN_BIAS_FWD";

	case CUDA_FN_SCALE_FWD:
		return "CUDA_FN_SCALE_FWD";

	case CUDA_FN_THRESHOLD_FWD:
		return "CUDA_FN_THRESHOLD_FWD";

	case CUDA_FN_CLL_BWD:
		return "CUDA_FN_CLL_BWD";

	case CUDA_FN_SMOOTHL1_FWD:
		return "CUDA_FN_SMOOTHL1_FWD";

	case CUDA_FN_SMOOTHL1_BWD:
		return "CUDA_FN_SMOOTHL1_BWD";

	case CUDA_FN_SERF_FWD:
		return "CUDA_FN_SERF_FWD";

	case CUDA_FN_SERF_BWD:
		return "CUDA_FN_SERF_BWD";

	case CUDA_FN_PERMUTE:
		return "CUDA_FN_PERMUTE";

	case CUDA_FN_GATHER_FWD:
		return "CUDA_FN_GATHER_FWD";

	case CUDA_FN_GATHER_BWD:
		return "CUDA_FN_GATHER_BWD";

	case CUDA_FN_LRN_FILLSCALE:
		return "CUDA_FN_LRN_FILLSCALE";

	case CUDA_FN_LRN_COMPUTEOUTPUT:
		return "CUDA_FN_LRN_COMPUTEOUTPUT";

	case CUDA_FN_LRN_COMPUTEDIFF:
		return "CUDA_FN_LRN_COMPUTEDIFF";

	case CUDA_FN_LSTM_FWD:
		return "CUDA_FN_LSTM_FWD";

	case CUDA_FN_LSTM_BWD:
		return "CUDA_FN_LSTM_BWD";

	case CUDA_FN_LSTM_UNIT_FWD:
		return "CUDA_FN_LSTM_UNIT_FWD";

	case CUDA_FN_LSTM_UNIT_BWD:
		return "CUDA_FN_LSTM_UNIT_BWD";

	case CUDA_FN_MATH_FWD:
		return "CUDA_FN_MATH_FWD";

	case CUDA_FN_MATH_BWD:
		return "CUDA_FN_MATH_BWD";

	case CUDA_FN_COEFF_SUM_FWD:
		return "CUDA_FN_COEFF_SUM_FWD";

	case CUDA_FN_COEFF_SUM_BWD:
		return "CUDA_FN_COEFF_SUM_BWD";

	case CUDA_FN_COEFF_SUB_FWD:
		return "CUDA_FN_COEFF_SUB_FWD";

	case CUDA_FN_COEFF_SUB_BWD:
		return "CUDA_FN_COEFF_SUB_BWD";

	case CUDA_FN_SIGMOID_CROSS_ENTROPY_FWD:
		return "CUDA_FN_SIGMOID_CROSS_ENTROPY_FWD";

	case CUDA_FN_SIGMOID_CROSS_ENTROPY_BWD:
		return "CUDA_FN_SIGMOID_CROSS_ENTROPY_BWD";

	case CUDA_FN_SOFTMAX_CROSS_ENTROPY_FWD:
		return "CUDA_FN_SOFTMAX_CROSS_ENTROPY_FWD";

	case CUDA_FN_SOFTMAX_CROSS_ENTROPY_BWD:
		return "CUDA_FN_SOFTMAX_CROSS_ENTROPY_BWD";

	case CUDA_FN_SGD_UPDATE:
		return "CUDA_FN_SGD_UPDATE";

	case CUDA_FN_NESTEROV_UPDATE:
		return "CUDA_FN_NESTEROV_UPDATE";

	case CUDA_FN_ADAGRAD_UPDATE:
		return "CUDA_FN_ADAGRAD_UPDATE";

	case CUDA_FN_ADADELTA_UPDATE:
		return "CUDA_FN_ADADELTA_UPDATE";

	case CUDA_FN_ADAM_UPDATE:
		return "CUDA_FN_ADAM_UPDATE";

	case CUDA_FN_ADAMW_UPDATE:
		return "CUDA_FN_ADAMW_UPDATE";

	case CUDA_FN_RMSPROP_UPDATE:
		return "CUDA_FN_RMSPROP_UPDATE";

	case CUDA_FN_COMBINE_DATA:
		return "CUDA_FN_COMBINE_DATA";

	case CUDA_FN_MTX_SET_DIAGONAL:
		return "CUDA_FN_MTX_SET_DIAGONAL";

	case CUDA_FN_MTX_SET_DIAGONAL2:
		return "CUDA_FN_MTX_SET_DIAGONAL2";

	case CUDA_FN_MTX_ADD_VECTOR:
		return "CUDA_FN_MTX_ADD_VECTOR";

	case CUDA_FN_MTX_TRANSPOSE_OP:
		return "CUDA_FN_MTX_TRANSPOSE_OP";

	case CUDA_FN_MTX_AGGREGATE_COLS:
		return "CUDA_FN_MTX_AGGREGATE_COLS";

	case CUDA_FN_MTX_AGGREGATE_ROWS:
		return "CUDA_FN_MTX_AGGREGATE_ROWS";

	case CUDA_FN_MTX_TRANSPOSE:
		return "CUDA_FN_MTX_TRANSPOSE";

	case CUDA_FN_MTX_MEANCENTER_BY_COL:
		return "CUDA_FN_MTX_MEANCENTER_BY_COL";

	case CUDA_FN_MTX_EUCLIDEAN_DIST:
		return "CUDA_FN_MTX_EUCLIDEAN_DIST";

	case CUDA_FN_MTX_DOT:
		return "CUDA_FN_MTX_DOT";

	case CUDA_FN_MTX_MEAN:
		return "CUDA_FN_MTX_MEAN";

	case CUDA_FN_MTX_STDEV:
		return "CUDA_FN_MTX_STDEV";

	case CUDA_FN_MTX_CORRELATION:
		return "CUDA_FN_MTX_CORRELATION";

	case CUDA_FN_TSNE_UPDATE:
		return "CUDA_FN_TSNE_UPDATE";

	case CUDA_FN_TSNE_UPDATE_GRAD:
		return "CUDA_FN_TSNE_UPDATE_GRAD";

	case CUDA_FN_TSNE_COMPUTE_EXACT_ERROR:
		return "CUDA_FN_TSNE_COMPUTE_EXACT_ERROR";

	case CUDA_FN_TSNE_COMPUTE_SQUARED_EUCLIDEAN_DISTANCE:
		return "CUDA_FN_TSNE_COMPUTE_SQUARED_EUCLIDEAN_DISTANCE";

	case CUDA_FN_TSNE_COMPUTE_Q_MATRIX:
		return "CUDA_FN_TSNE_COMPUTE_Q_MATRIX";

	case CUDA_FN_TSNE_COMPUTE_EXACT_GRADIENT:
		return "CUDA_FN_TSNE_COMPUTE_EXACT_GRADIENT";

	case CUDA_FN_TSNE_SYMMETRIZE_MATRIX:
		return "CUDA_FN_TSNE_SYMMETRIZE_MATRIX";

	case CUDA_FN_TSNE_COMPUTE_KNN_BOUNDS:
		return "CUDA_FN_TSNE_COMPUTE_KNN_BOUNDS";

	case CUDA_FN_GUASSIAN_BLUR:
		return "CUDA_FN_GUASSIAN_BLUR";

	case CUDA_FN_HAMMING_DIFF:
		return "CUDA_FN_HAMMING_DIFF";

	case CUDA_FN_CALC_BATCH_DIST:
		return "CUDA_FN_CALC_BATCH_DIST";

	case CUDA_FN_CREATE_SSD:
		return "CUDA_FN_CREATE_SSD";

	case CUDA_FN_FREE_SSD:
		return "CUDA_FN_FREE_SSD";

	case CUDA_FN_SETUP_SSD:
		return "CUDA_FN_SETUP_SSD";

	case CUDA_FN_SSD_FWD_MULTIBOXLOSS:
		return "CUDA_FN_SSD_FWD_MULTIBOXLOSS";

	case CUDA_FN_SSD_ENCODE_LOCPRED:
		return "CUDA_FN_SSD_ENCODE_LOCPRED";

	case CUDA_FN_SSD_ENCODE_CONFPRED:
		return "CUDA_FN_SSD_ENCODE_CONFPRED";

	case CUDA_FN_CREATE_LAYERNORM:
		return "CUDA_FN_CREATE_LAYERNORM";

	case CUDA_FN_FREE_LAYERNORM:
		return "CUDA_FN_FREE_LAYERNORM";

	case CUDA_FN_LAYERNORM_FWD:
		return "CUDA_FN_LAYERNORM_FWD";

	case CUDA_FN_LAYERNORM_BWD:
		return "CUDA_FN_LAYERNORM_BWD";

	default:
		return "UNKNOWN";
	}
}
#endif
#endif

template <class T>
long Kernel<T>::Query(long lfnIdx, LONG* pfInput, long lCount, LPTSTR* ppOutput)
{
	hipGetLastError();

	switch (lfnIdx)
	{
		case CUDA_FN_GET_DEVICE_NAME:
			return m_device.GetDeviceName(lCount, pfInput, ppOutput);

		case CUDA_FN_GET_P2P_INFO:
			return m_device.GetDeviceP2PInfo(lCount, pfInput, ppOutput);

		case CUDA_FN_GET_DEVICE_INFO:
			return m_device.GetDeviceInfo(lCount, pfInput, ppOutput);

		default:
			return ERROR_PARAM_OUT_OF_RANGE;
	}
}

template long Kernel<double>::Query(long lfnIdx, LONG* pfInput, long lCount, LPTSTR* ppOutput);
template long Kernel<float>::Query(long lfnIdx, LONG* pfInput, long lCount, LPTSTR* ppOutput);


//end main.cu