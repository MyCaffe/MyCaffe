#include "hip/hip_runtime.h"
//=============================================================================
//	FILE:	util.cu
//
//	DESC:	This file implements the utility functions.
//=============================================================================

#include "util.h"

//=============================================================================
//	Constants
//=============================================================================

//=============================================================================
//	Public Methods
//=============================================================================

inline double cint(double x)
{
	double dfInt = 0;

	if (modf(x, &dfInt) >= 0.5)
		return (x >= 0) ? ceil(x) : floor(x);
	else
		return (x < 0) ? ceil(x) : floor(x);
}

inline double round2(double r, int places)
{
	double off = pow(10.0, places);
	return cint(r*off)/off;
}

inline double roundex(double r)
{
	char sz[256];

	sprintf(sz, "%0.7lf", r);
	return atof(sz);
}

bool GetErrorString(long lErr, char* szErr, long lMaxErr)
{
	if (GetCudaErrorString(lErr, szErr, lMaxErr))
		return true;

	switch (lErr)
	{
		case ERROR_PARAM:
			_snprintf(szErr, lMaxErr, "GENERAL: Parameter error (%ld)", lErr);
			return true;

		case ERROR_PARAM_OUT_OF_RANGE:
			_snprintf(szErr, lMaxErr, "GENERAL: Parameter out of range (%ld)", lErr);
			return true;

		case ERROR_PARAM_NULL:
			_snprintf(szErr, lMaxErr, "GENERAL: Parameter is NULL (%ld)", lErr);
			return true;

		case ERROR_FILE_WRITE:
			_snprintf(szErr, lMaxErr, "GENERAL: Failure when writing to file (%ld)", lErr);
			return true;

		case ERROR_FILE_READ:
			_snprintf(szErr, lMaxErr, "GENERAL: Failure when reading from file (%ld)", lErr);
			return true;

		case ERROR_FILE_OPEN:
			_snprintf(szErr, lMaxErr, "GENERAL: Failure when opening a file (%ld)", lErr);
			return true;

		case ERROR_MATRIX:
			_snprintf(szErr, lMaxErr, "MATRIX: general matrix error (%ld)", lErr);
			return true;

		case ERROR_MEMORY_EXPECTED_DEVICE:
			_snprintf(szErr, lMaxErr, "MEMORY: Expected device memory but received host memory (%ld)", lErr);
			return true;

		case ERROR_MEMORY_RANGE_EXCEEDED:
			_snprintf(szErr, lMaxErr, "MEMORY: Exceeded the maximum amount of memory size available as a chunk (%ld)", lErr);
			return true;

		case ERROR_MEMORY_MIXED_HALF_TYPES:
			_snprintf(szErr, lMaxErr, "MEMORY: You are using a mix of half types and non-half types.  All types for this function must be of the same type (%ld)", lErr);
			return true;

		case ERROR_MEMORY_HALF_TYPE_NOT_SUPPORTED:
			_snprintf(szErr, lMaxErr, "MEMORY: The GPU that you are using has limited half-type support.  Full half-type support is only available on Maxwell gpu's with compute 5.3 and above (%ld)", lErr);
			return true;

		case ERROR_MEMORY_OUT:
			_snprintf(szErr, lMaxErr, "MEMORY: Out of memory (%ld)", lErr);
			return true;

		case ERROR_MATRIX_DIMENSIONS_DONT_MATCH:
			_snprintf(szErr, lMaxErr, "MATRIX: matrix dimensions do not match (%ld)", lErr);
			return true;

		case ERROR_MATRIX_DIMENSIONS_EXCEED_THREADS:
			_snprintf(szErr, lMaxErr, "MATRIX: matrix dimensions exceed number of threads (%ld)", lErr);
			return true;

		case ERROR_MATRIX_NOT_SQUARE:
			_snprintf(szErr, lMaxErr, "MATRIX: the current operation is only supported on square matrices (%ld)", lErr);
			return true;

		case ERROR_VECTOR:
			_snprintf(szErr, lMaxErr, "VECTOR: general vector error (%ld)", lErr);
			return true;

		case ERROR_VECTOR_DIMENSIONS_DONT_MATCH:
			_snprintf(szErr, lMaxErr, "VECTOR: vector dimensions do not match (%ld)", lErr);
			return true;

		case ERROR_NN:
			_snprintf(szErr, lMaxErr, "NN: general neural net error (%ld)", lErr);
			return true;

		case ERROR_NN_LAYER_COUNTS_DONT_MATCH:
			_snprintf(szErr, lMaxErr, "NN: layer counts do not match (%ld)", lErr);
			return true;

		case ERROR_CUBLAS_NULL:
			_snprintf(szErr, lMaxErr, "NN: The cublas handle is NULL! (%ld)", lErr);
			return true;

		case ERROR_CUDA_NOTSUPPORED_ON_DISPLAYGPU:
			_snprintf(szErr, lMaxErr, "CUDA: The function you are attempting to run is not supported on the display GPU (only supported on headless gpus)! (%ld)", lErr);
			return true;

		case ERROR_CUDA_MISSING_NCCL64DLL:
			_snprintf(szErr, lMaxErr, "CUDA: The 'nccl64' DLL is missing from the executable directory!  For example when using the version 134 for CUDA 10.0, the file 'nccl64_134.10.0.dll' should be in the same directory as the executable. (%ld)", lErr);
			return true;

		case ERROR_TSNE:
			_snprintf(szErr, lMaxErr, "TSNE: A general TSN-E error occurred. (%ld)", lErr);
			return true;

		case ERROR_TSNE_NO_DISTANCES_FOUND:
			_snprintf(szErr, lMaxErr, "TSNE: No differences found between the images - they may all be the same. (%ld)", lErr);
			return true;

		case ERROR_SSD:
			_snprintf(szErr, lMaxErr, "SSD: A general SSD error occurred. (%ld)", lErr);
			return true;

		case ERROR_SSD_NOT_INITIALIZED:
			_snprintf(szErr, lMaxErr, "SSD: The SSD is not initialized. (%ld)", lErr);
			return true;

		case ERROR_SSD_INVALID_CODE_TYPE:
			_snprintf(szErr, lMaxErr, "SSD: The SSD code type specified is invalid. (%ld)", lErr);
			return true;

		case ERROR_SSD_INVALID_BBOX_DIMENSION:
			_snprintf(szErr, lMaxErr, "SSD: The SSD bbox dimension (width or height) is invalid (e.g. < 0). (%ld)", lErr);
			return true;

		case ERROR_SSD_HOST_TYPE_NOT_SUPPORTED:
			_snprintf(szErr, lMaxErr, "SSD: The HOST type specified is not supported for this function. (%ld)", lErr);
			return true;

		case ERROR_SSD_BAD_MATCH:
			_snprintf(szErr, lMaxErr, "SSD: The current matching is bad, expected a match index of -1. (%ld)", lErr);
			return true;

		case ERROR_SSD_GT_LABEL_OUT_OF_RANGE:
			_snprintf(szErr, lMaxErr, "SSD: The ground truth label is out of range. (%ld)", lErr);
			return true;

		case ERROR_SSD_PRIOR_VARIANCE_COUNT:
			_snprintf(szErr, lMaxErr, "SSD: The prior variances count does not match the prior bbox count. (%ld)", lErr);
			return true;

		case ERROR_SSD_BACKGROUND_LABEL_OUT_OF_RANGE:
			_snprintf(szErr, lMaxErr, "SSD: The background label id is out of range. (%ld)", lErr);
			return true;

		case ERROR_SSD_COMPUTE_CONF_LOSS_MATCH_INDEX_INCORRECT:
			_snprintf(szErr, lMaxErr, "SSD: The match_index should equal the number of priors in the compute conf loss calculation. (%ld)", lErr);
			return true;

		case ERROR_SSD_COMPUTE_CONF_LOSS_GT_MISSING_ITEM:
			_snprintf(szErr, lMaxErr, "SSD: The ground-truths are missing an expected itemId in the compute conf loss calculation. (%ld)", lErr);
			return true;

		case ERROR_SSD_COMPUTE_CONF_LOSS_MATCH_INDEX_OUT_OF_RANGE:
			_snprintf(szErr, lMaxErr, "SSD: The match index is out of range of the ground-truths in the compute conf loss calculation. (%ld)", lErr);
			return true;

		case ERROR_SSD_COMPUTE_CONF_LOSS_INVALID_LABEL:
			_snprintf(szErr, lMaxErr, "SSD: The label in the compute conf loss calculation is invalid. (%ld)", lErr);
			return true;

		case ERROR_SSD_NOT_SUPPORTED_IN_HALF_BBOX:
			_snprintf(szErr, lMaxErr, "SSD: The requested query is not supported by the half Bbox - only full BBox's support this type of query. (%ld)", lErr);
			return true;

		case ERROR_SSD_LOC_PRED_LABEL_NOT_FOUND:
			_snprintf(szErr, lMaxErr, "SSD: Could not find an expected label in the loc predictions. (%ld)", lErr);
			return true;

		case ERROR_SSD_CODE_TYPE_UNKNOWN:
			_snprintf(szErr, lMaxErr, "SSD: Unknown Code-Type. (%ld)", lErr);
			return true;

		case ERROR_SSD_SAMPLE_SIZE_TOO_SMALL:
			_snprintf(szErr, lMaxErr, "SSD: The sample size is too small and must be > 0. (%ld)", lErr);
			return true;

		case ERROR_SSD_INVALID_NUM_CLASSES:
			_snprintf(szErr, lMaxErr, "SSD: The number of classes is incorrect (e.g. when using map to agnostic, only 2 classes are valid for backgroundLabel >= 0, otherwise only 1 class is valid). (%ld)", lErr);
			return true;

		case ERROR_SSD_CONF_LOSS_TYPE_UNKNOWN:
			_snprintf(szErr, lMaxErr, "SSD: The conf loss type is unknown and invalid. (%ld)", lErr);
			return true;

		case ERROR_SSD_BACKGROUND_LABEL_IN_DATASET:
			_snprintf(szErr, lMaxErr, "SSD: The ground truth was found in the dataset. (%ld)", lErr);
			return true;

		case ERROR_SSD_NUMLOCCLASSES_INVALID_FOR_SHARED:
			_snprintf(szErr, lMaxErr, "SSD: The number of loc classes must be 1 when using shared location. (%ld)", lErr);
			return true;
	}

	return false;
}

bool GetCudaErrorString(long lErr, char* szErr, long lMaxErr)
{
	if (lErr == 0)
		return false;

	if ((lErr & ERROR_CUBLAS_OFFSET) == ERROR_CUBLAS_OFFSET)
	{
		lErr &= (~ERROR_CUBLAS_OFFSET);

		switch (lErr)
		{
		case HIPBLAS_STATUS_NOT_INITIALIZED:
			_snprintf(szErr, lMaxErr, "cuBlas: The cuBlas library was not initialized propertly (%ld)", lErr);
			return true;

		case HIPBLAS_STATUS_ALLOC_FAILED:
			_snprintf(szErr, lMaxErr, "cuBlas: A resource allocation failed within the cuBlas library (%ld)", lErr);
			return true;

		case HIPBLAS_STATUS_INVALID_VALUE:
			_snprintf(szErr, lMaxErr, "cuBlas: An invalid parameter was passed to the function. (%ld)", lErr);
			return true;

		case HIPBLAS_STATUS_ARCH_MISMATCH:
			_snprintf(szErr, lMaxErr, "cuBlas: The function requires functionality not supported by the current device architecture. (%ld)", lErr);
			return true;

		case HIPBLAS_STATUS_MAPPING_ERROR:
			_snprintf(szErr, lMaxErr, "cuBlas: Access to the GPU memory failed possibly caused by a failure to bind to a texture. (%ld)", lErr);
			return true;

		case HIPBLAS_STATUS_EXECUTION_FAILED:
			_snprintf(szErr, lMaxErr, "cuBlas: A cuBlas GPU kernel failed to execute. (%ld)", lErr);
			return true;

		case HIPBLAS_STATUS_INTERNAL_ERROR:
			_snprintf(szErr, lMaxErr, "cuBlas: A failure occurred within cuBlas. (%ld)", lErr);
			return true;

		case HIPBLAS_STATUS_NOT_SUPPORTED:
			_snprintf(szErr, lMaxErr, "cuBlas: The function called is not supported. (%ld)", lErr);
			return true;

		case HIPBLAS_STATUS_UNKNOWN:
			_snprintf(szErr, lMaxErr, "cuBlas: The functionality requested requires a license that is missing. (%ld)", lErr);
			return true;
		}
	}
	else if ((lErr & ERROR_CUDNN_OFFSET) == ERROR_CUDNN_OFFSET)
	{
		lErr &= (~ERROR_CUDNN_OFFSET);

		switch (lErr)
		{
			case HIPDNN_STATUS_NOT_INITIALIZED:
				_snprintf(szErr, lMaxErr, "cuDNN: The cuDNN library was not initialized propertly (%ld)", lErr);
				return true;

			case HIPDNN_STATUS_ALLOC_FAILED:
				_snprintf(szErr, lMaxErr, "cuDNN: A resource allocation failed within the cuDNN library (%ld)", lErr);
				return true;

			case HIPDNN_STATUS_BAD_PARAM:
				_snprintf(szErr, lMaxErr, "cuDNN: An incorrect parameter was passed to a function (%ld)", lErr);
				return true;

			case HIPDNN_STATUS_INTERNAL_ERROR:
				_snprintf(szErr, lMaxErr, "cuDNN: An internal operation failed (%ld)", lErr);
				return true;

			case HIPDNN_STATUS_INVALID_VALUE:
				_snprintf(szErr, lMaxErr, "cuDNN: An invalid value was detected (%ld)", lErr);
				return true;

			case HIPDNN_STATUS_ARCH_MISMATCH:
				_snprintf(szErr, lMaxErr, "cuDNN: The function requires a feature not supported by the current GPU device - your device must have compute capability of 3.0 or greater (%ld)", lErr);
				return true;

			case HIPDNN_STATUS_MAPPING_ERROR:
				_snprintf(szErr, lMaxErr, "cuDNN: An access to the GPU's memory space failed perhaps caused when binding to a texture (%ld)", lErr);
				return true;

			case HIPDNN_STATUS_EXECUTION_FAILED:
				_snprintf(szErr, lMaxErr, "cuDNN: The current GPU program failed to execute (%ld)", lErr);
				return true;

			case HIPDNN_STATUS_NOT_SUPPORTED:
				_snprintf(szErr, lMaxErr, "cuDNN: The functionality requested is not supported by this version of cuDNN (%ld)", lErr);
				return true;

			case HIPDNN_STATUS_LICENSE_ERROR:
				_snprintf(szErr, lMaxErr, "cuDNN: The functionality requested requires a license that does not appear to exist (%ld)", lErr);
				return true;

			case HIPDNN_STATUS_RUNTIME_PREREQUISITE_MISSING:
				_snprintf(szErr, lMaxErr, "cuDNN: The runtime library required by RNN calls (nvcuda.dll) cannot be found (%ld)", lErr);
				return true;

#if CUDNN_MAJOR >= 7
			case CUDNN_STATUS_RUNTIME_IN_PROGRESS:
				_snprintf(szErr, lMaxErr, "cuDNN: Some tasks in the user stream are still running (%ld)", lErr);
				return true;

			case CUDNN_STATUS_RUNTIME_FP_OVERFLOW:
				_snprintf(szErr, lMaxErr, "cuDNN: A numerical overflow occurred while executing the GPU kernel (%ld)", lErr);
				return true;
#endif
		}

		return false;
	}

	switch (lErr)
	{
		case hipErrorMissingConfiguration:
			_snprintf(szErr, lMaxErr, "CUDA: Missing configuration error (%ld)", lErr);
			return true;
			
		case hipErrorOutOfMemory:
			_snprintf(szErr, lMaxErr, "CUDA: Memory allocation error (%ld)", lErr);
			return true;
			
		case hipErrorNotInitialized:
			_snprintf(szErr, lMaxErr, "CUDA: Initialization error (%ld)", lErr);
			return true;
			
		case hipErrorLaunchFailure:
			_snprintf(szErr, lMaxErr, "CUDA: Launch failure (%ld)", lErr);
			return true;
			
		case hipErrorPriorLaunchFailure:
			_snprintf(szErr, lMaxErr, "CUDA: Prior launch failure (%ld)", lErr);
			return true;
			
		case hipErrorLaunchTimeOut:
			_snprintf(szErr, lMaxErr, "CUDA: Prior launch failure - timeout (%ld)", lErr);
			return true;
			
		case hipErrorLaunchOutOfResources:
			_snprintf(szErr, lMaxErr, "CUDA: Launch out of resources error (%ld)", lErr);
			return true;
			
		case hipErrorInvalidDeviceFunction:
			_snprintf(szErr, lMaxErr, "CUDA: Invalid device function (%ld)", lErr);
			return true;
			
		case hipErrorInvalidConfiguration:
			_snprintf(szErr, lMaxErr, "CUDA: Invalid configuration for the device used (%ld)", lErr);
			return true;
			
		case hipErrorInvalidDevice:
			_snprintf(szErr, lMaxErr, "CUDA: Invalid CUDA device (%ld)", lErr);
			return true;
			
		case hipErrorInvalidValue:
			_snprintf(szErr, lMaxErr, "CUDA: Invalid parameter value (%ld)", lErr);
			return true;
			
		case hipErrorInvalidPitchValue:
			_snprintf(szErr, lMaxErr, "CUDA: Invalid pitch parameter value (%ld)", lErr);
			return true;
			
		case hipErrorInvalidSymbol:
			_snprintf(szErr, lMaxErr, "CUDA: Invalid symbol (%ld)", lErr);
			return true;
			
		case hipErrorMapFailed:
			_snprintf(szErr, lMaxErr, "CUDA: Map buffer object failed (%ld)", lErr);
			return true;
			
		case hipErrorUnmapFailed:
			_snprintf(szErr, lMaxErr, "CUDA: Unmap buffer object failed (%ld)", lErr);
			return true;
			
		case cudaErrorInvalidHostPointer:
			_snprintf(szErr, lMaxErr, "CUDA: Invalid host pointer (%ld)", lErr);
			return true;
			
		case hipErrorInvalidDevicePointer:
			_snprintf(szErr, lMaxErr, "CUDA: Invalid device pointer (%ld)", lErr);
			return true;
			
		case hipErrorInvalidTexture:
			_snprintf(szErr, lMaxErr, "CUDA: Invalid texture (%ld)", lErr);
			return true;
			
		case cudaErrorInvalidTextureBinding:
			_snprintf(szErr, lMaxErr, "CUDA: Invalid texture binding (%ld)", lErr);
			return true;
			
		case hipErrorInvalidChannelDescriptor:
			_snprintf(szErr, lMaxErr, "CUDA: Invalid channel descriptor (%ld)", lErr);
			return true;
			
		case hipErrorInvalidMemcpyDirection:
			_snprintf(szErr, lMaxErr, "CUDA: Invalid memcpy direction (%ld)", lErr);
			return true;
			
		case cudaErrorAddressOfConstant:
			_snprintf(szErr, lMaxErr, "CUDA: Address of constant error (%ld)", lErr);
			return true;
			
		case cudaErrorTextureFetchFailed:
			_snprintf(szErr, lMaxErr, "CUDA: Texture fetch failed (%ld)", lErr);
			return true;
			
		case cudaErrorTextureNotBound:
			_snprintf(szErr, lMaxErr, "CUDA: Texture not bound error (%ld)", lErr);
			return true;
			
		case cudaErrorSynchronizationError:
			_snprintf(szErr, lMaxErr, "CUDA: Synchronization error (%ld)", lErr);
			return true;
			
		case cudaErrorInvalidFilterSetting:
			_snprintf(szErr, lMaxErr, "CUDA: Invalid filter setting (%ld)", lErr);
			return true;
			
		case cudaErrorInvalidNormSetting:
			_snprintf(szErr, lMaxErr, "CUDA: Invalid norm setting (%ld)", lErr);
			return true;
			
		case cudaErrorMixedDeviceExecution:
			_snprintf(szErr, lMaxErr, "CUDA: Mixed device execution (%ld)", lErr);
			return true;
			
		case hipErrorDeinitialized:
			_snprintf(szErr, lMaxErr, "CUDA: cuda runtime unloading (%ld)", lErr);
			return true;
			
		case hipErrorUnknown:
			_snprintf(szErr, lMaxErr, "CUDA: Unknown error condition (%ld)", lErr);
			return true;
			
		case cudaErrorNotYetImplemented:
			_snprintf(szErr, lMaxErr, "CUDA: Function not yet implemented (%ld)", lErr);
			return true;
			
		case cudaErrorMemoryValueTooLarge:
			_snprintf(szErr, lMaxErr, "CUDA: Memory value too large (%ld)", lErr);
			return true;
			
		case hipErrorInvalidHandle:
			_snprintf(szErr, lMaxErr, "CUDA: Invalid resource handle (%ld)", lErr);
			return true;
			
		case hipErrorNotReady:
			_snprintf(szErr, lMaxErr, "CUDA: Not ready error (%ld)", lErr);
			return true;
			
		case hipErrorInsufficientDriver:
			_snprintf(szErr, lMaxErr, "CUDA: cuda runtime is newer than the installed NVIDIA CUDA driver (%ld)", lErr);
			return true;
			
		case hipErrorSetOnActiveProcess:
			_snprintf(szErr, lMaxErr, "CUDA: Set on active process error (%ld)", lErr);
			return true;

		case cudaErrorInvalidSurface:
			_snprintf(szErr, lMaxErr, "CUDA: Indicates that the surface parameter is invalid (%ld)", lErr);
			return true;

		case hipErrorNoDevice:
			_snprintf(szErr, lMaxErr, "CUDA: No available CUDA device (%ld)", lErr);
			return true;
			
		case hipErrorECCNotCorrectable:
			_snprintf(szErr, lMaxErr, "CUDA: Uncorrectable ECC error detected (%ld)", lErr);
			return true;

		case hipErrorSharedObjectSymbolNotFound:
			_snprintf(szErr, lMaxErr, "CUDA: The link to to a shared object failed to resolve (%ld)", lErr);
			return true;

		case hipErrorSharedObjectInitFailed:
			_snprintf(szErr, lMaxErr, "CUDA: The initialization of a shared object failed (%ld)", lErr);
			return true;

		case hipErrorUnsupportedLimit:
			_snprintf(szErr, lMaxErr, "CUDA: The ::hipLimit_t argument is not supported by the active device (%ld)", lErr);
			return true;

		case cudaErrorDuplicateVariableName:
			_snprintf(szErr, lMaxErr, "CUDA: Inidcates that multiple global or constant variables share the same string name (%ld)", lErr);
			return true;

		case cudaErrorDuplicateTextureName:
			_snprintf(szErr, lMaxErr, "CUDA: Inidcates that multiple texture variables share the same string name (%ld)", lErr);
			return true;

		case cudaErrorDuplicateSurfaceName:
			_snprintf(szErr, lMaxErr, "CUDA: Inidcates that multiple surface variables share the same string name (%ld)", lErr);
			return true;

		case cudaErrorDevicesUnavailable:
			_snprintf(szErr, lMaxErr, "CUDA: Indicates that all CUDA devices are busy or unavailable at the current time (%ld)", lErr);
			return true;

		case hipErrorInvalidImage:
			_snprintf(szErr, lMaxErr, "CUDA: Indicates that the device kernel image is invalid (%ld)", lErr);
			return true;

		case hipErrorNoBinaryForGpu:
			_snprintf(szErr, lMaxErr, "CUDA: Indicates that there is no kernel image available that is suitable for the device (%ld)", lErr);
			return true;

		case cudaErrorIncompatibleDriverContext:
			_snprintf(szErr, lMaxErr, "CUDA: Indicates that the current context is not compatible with this CUDA Runtime (%ld)", lErr);
			return true;

		case hipErrorPeerAccessAlreadyEnabled:
			_snprintf(szErr, lMaxErr, "CUDA: Indicates that a call to ::hipDeviceEnablePeerAccess is trying to re-enable peer addressing from a context that already has peer addressing enabled (%ld)", lErr);
			return true;

		case hipErrorPeerAccessNotEnabled:
			_snprintf(szErr, lMaxErr, "CUDA: Indicates that ::hipDeviceDisablePeerAccess is trying to disable peer addressing which has not been enabled yet (%ld)", lErr);
			return true;

		case hipErrorContextAlreadyInUse:
			_snprintf(szErr, lMaxErr, "CUDA: Indicates that a call tried to access an exclusive-thread device that is already in use by a different thread (%ld)", lErr);
			return true;

		case hipErrorProfilerDisabled:
			_snprintf(szErr, lMaxErr, "CUDA: Indicates profiler is not initialized for this run (%ld)", lErr);
			return true;

		case hipErrorAssert:
			_snprintf(szErr, lMaxErr, "CUDA: An assert triggered in device code during kernel execution (%ld)", lErr);
			return true;

		case cudaErrorTooManyPeers:
			_snprintf(szErr, lMaxErr, "CUDA: Indicates that the hardware resources required ot enable peer access have been exhaused for one or more of the devices (%ld)", lErr);
			return true;

		case hipErrorHostMemoryAlreadyRegistered:
			_snprintf(szErr, lMaxErr, "CUDA: Indicates that the memory range specified has already been registered (%ld)", lErr);
			return true;

		case hipErrorHostMemoryNotRegistered:
			_snprintf(szErr, lMaxErr, "CUDA: Indicates that the pointer specified does not correspond to any currently registered memory region (%ld)", lErr);
			return true;

		case hipErrorOperatingSystem:
			_snprintf(szErr, lMaxErr, "CUDA: Indicates that an OS call failed (%ld)", lErr);
			return true;

		case hipErrorPeerAccessUnsupported:
			_snprintf(szErr, lMaxErr, "CUDA: Indicates that P2P access is not supported across the given devices (%ld)", lErr);
			return true;

		case cudaErrorLaunchMaxDepthExceeded:
			_snprintf(szErr, lMaxErr, "CUDA: Indicates that a device runtime grid launch did not occur because  the depth of the child grid would exceed the maximum supported number of nested grid launches (%ld)", lErr);
			return true;

		case cudaErrorLaunchFileScopedTex:
			_snprintf(szErr, lMaxErr, "CUDA: Indicates that a grid launch did no occur because the kernel uses file-scoped textures which are unsupported by the device runtime (%ld)", lErr);
			return true;

		case cudaErrorLaunchFileScopedSurf:
			_snprintf(szErr, lMaxErr, "CUDA: Indicates that a grid launch did not occur because the kernel uses file-scoped surfaces which are unsupported by the device runtime. (%ld)", lErr);
			return true;

		case cudaErrorSyncDepthExceeded:
			_snprintf(szErr, lMaxErr, "CUDA: Indicates that a call to ::hipDeviceSynchronize made from the device runtime failed becaue the call was made at grid depth greater than either the default (2 levels) or a user specified limit (%ld)", lErr);
			return true;

		case cudaErrorLaunchPendingCountExceeded:
			_snprintf(szErr, lMaxErr, "CUDA: Indicates that a device runtime grid launch failed because the launch would exceed the limit ::cudaLimitDevRuntimePendingLaunchCount (%ld)", lErr);
			return true;

		case cudaErrorNotPermitted:
			_snprintf(szErr, lMaxErr, "CUDA: Indicates the attempted operation is not permitted (%ld)", lErr);
			return true;

		case hipErrorNotSupported:
			_snprintf(szErr, lMaxErr, "CUDA: Indicates the attempted operation is not supported on the current system or device (%ld)", lErr);
			return true;

		case cudaErrorHardwareStackError:
			_snprintf(szErr, lMaxErr, "CUDA: Device encountered an error in the call statck during kernel execution possibly due to stack corruption or exceeding the stack size limit (%ld)", lErr);
			return true;

		case cudaErrorIllegalInstruction:
			_snprintf(szErr, lMaxErr, "CUDA: Device encountered an illegal instruction during kernel execution (%ld)", lErr);
			return true;

		case cudaErrorMisalignedAddress:
			_snprintf(szErr, lMaxErr, "CUDA: Device encountered a load or storage instruction on a memory address which is not aligned (%ld)", lErr);
			return true;

		case cudaErrorInvalidAddressSpace:
			_snprintf(szErr, lMaxErr, "CUDA: While executing a kernel, the device encountered an instruction which can only operate on memory locations in certain address spaces (global, shared, or local), but was supplied an address not in those spaces (%ld)", lErr);
			return true;

		case cudaErrorInvalidPc:
			_snprintf(szErr, lMaxErr, "CUDA: Device encountered an invalid program counter (%ld)", lErr);
			return true;

		case hipErrorIllegalAddress:
			_snprintf(szErr, lMaxErr, "CUDA: Device encountered a load or storage instruction on an invalid memory address (%ld)", lErr);
			return true;

		case hipErrorInvalidKernelFile:
			_snprintf(szErr, lMaxErr, "CUDA: A PTX compilation failed (%ld)", lErr);
			return true;

		case hipErrorInvalidGraphicsContext:
			_snprintf(szErr, lMaxErr, "CUDA: Indicates an error with the OpenGL or DirectX context (%ld)", lErr);
			return true;

		case cudaErrorNvlinkUncorrectable:
			_snprintf(szErr, lMaxErr, "CUDA: Indicates an uncorrectable NVLink error was detected during the execution (%ld)", lErr);
			return true;

		case cudaErrorStartupFailure:
			_snprintf(szErr, lMaxErr, "CUDA: Startup failure (%ld)", lErr);
			return true;
	}

	return false;
}


//=============================================================================
//	Device Functions
//=============================================================================

//end util.cu