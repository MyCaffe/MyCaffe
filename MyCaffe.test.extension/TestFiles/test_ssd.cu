#include "hip/hip_runtime.h"
//=============================================================================
//	FILE:	test_ssd.cu
//
//	DESC:	This file implements the single-shot multi-box detection testing code.
//=============================================================================

#include "..\Cuda Files\util.h"
#include "test_ssd.h"

#include "..\Cuda Files\memory.h"
#include "..\Cuda Files\math.h"
#include "..\Cuda Files\ssd.h"


//=============================================================================
//	Test Enum.
//=============================================================================

enum TEST
{
	CREATE = 1,

	BBOX_SIZE = 2,
	BBOX_BOUNDS = 3,
	BBOX_DIVBOUNDS = 4,
	BBOX_CLIP = 5,

	BBOX_DECODE = 6,
	BBOX_ENCODE = 7,
	BBOX_INTERSECT = 8,
	BBOX_JACCARDOVERLAP = 9,
	BBOX_MATCH = 10,

	FINDMATCHES = 11,
	COUNTMATCHES = 12,
	SOFTMAX = 13,
	COMPUTE_CONF_LOSS = 14,
	COMPUTE_LOC_LOSS = 15,
	GET_TOPK_SCORES = 16,
	APPLYNMS = 17,
	MINE_HARD_EXAMPLES = 18
};


//=============================================================================
//	Test Helper Classes
//=============================================================================

template <class T>
class TestData
{
	Memory<T> m_memory;
	Math<T> m_math;
	T m_fEps;

public:
	SsdData<T> m_ssd;

	TestData() : m_memory(), m_math(), m_ssd(&m_memory, &m_math)
	{
		m_fEps = (T)1e-6;
	}

	void EXPECT_NEAR(T t1, T t2, T fErr = 0)
	{
		if (fErr == 0)
			fErr = m_fEps;

		T fDiff = (T)fabs(t1 - t2);
		if (fDiff > fErr)
			throw ERROR_PARAM_OUT_OF_RANGE;
	}

	long TestCreate(int nConfig)
	{
		LONG lErr;

		if (lErr = m_ssd.Initialize(0, 2, true, 2, 0, false, SSD_MINING_TYPE_NONE, SSD_MATCHING_TYPE_BIPARTITE, 0.3, true, SSD_CODE_TYPE_CORNER, true, false, true, true, SSD_CONF_LOSS_TYPE_SOFTMAX, SSD_LOC_LOSS_TYPE_L2, 0, 0, 10, false, 0.1, 10, 0.1))
			return lErr;

		return 0;
	}

	long TestBBOX_Size(int nConfig)
	{
		T fSize;

		// Valid box.
		T xmin = T(0.2);
		T ymin = T(0.3);
		T xmax = T(0.3);
		T ymax = T(0.5);
		fSize = SsdBbox<T>::getSize(xmin, ymin, xmax, ymax, true);
		EXPECT_NEAR(fSize, T(0.02));

		// A line.
		xmin = T(0.2);
		ymin = T(0.3);
		xmax = T(0.2);
		ymax = T(0.5);
		fSize = SsdBbox<T>::getSize(xmin, ymin, xmax, ymax, true);
		EXPECT_NEAR(fSize, T(0.0));

		// Invalid box.
		xmin = T(0.2);
		ymin = T(0.3);
		xmax = T(0.1);
		ymax = T(0.5);
		fSize = SsdBbox<T>::getSize(xmin, ymin, xmax, ymax, true);
		EXPECT_NEAR(fSize, T(0.0));

		return 0;
	}

	long TestBBOX_Bounds(int nConfig)
	{
		return ERROR_NOT_IMPLEMENTED;
	}

	long TestBBOX_DivBounds(int nConfig)
	{
		return ERROR_NOT_IMPLEMENTED;
	}

	long TestBBOX_Clip(int nConfig)
	{
		T xmin = T(0.2);
		T ymin = T(0.3);
		T xmax = T(0.3);
		T ymax = T(0.5);

		T xmin2 = T(0.2);
		T ymin2 = T(0.3);
		T xmax2 = T(0.3);
		T ymax2 = T(0.5);

		SsdBbox<T>::clip(&xmin, &ymin, &xmax, &ymax);
		T fSize = SsdBbox<T>::getSize(xmin, ymin, xmax, ymax);
		EXPECT_NEAR(xmin, xmin2);
		EXPECT_NEAR(ymin, ymin2);
		EXPECT_NEAR(xmax, xmax2);
		EXPECT_NEAR(ymax, ymax2);
		EXPECT_NEAR(fSize, T(0.02));

		xmin = T(-0.2);
		ymin = T(-0.3);
		xmax = T(1.3);
		ymax = T(1.5);
		SsdBbox<T>::clip(&xmin, &ymin, &xmax, &ymax);
		fSize = SsdBbox<T>::getSize(xmin, ymin, xmax, ymax);
		EXPECT_NEAR(xmin, T(0.0));
		EXPECT_NEAR(ymin, T(0.0));
		EXPECT_NEAR(xmax, T(1.0));
		EXPECT_NEAR(ymax, T(1.0));
		EXPECT_NEAR(fSize, T(1.0));

		fSize = SsdBbox<T>::getSize(xmin, ymin, xmax, ymax);
		EXPECT_NEAR(fSize, T(1.0));

		return 0;
	}

	long TestBBOX_Decode(int nConfig)
	{
		return ERROR_NOT_IMPLEMENTED;
	}

	long TestBBOX_Encode(int nConfig)
	{
		return ERROR_NOT_IMPLEMENTED;
	}

	long TestBBOX_Intersect(int nConfig)
	{
		T xmin;
		T ymin;
		T xmax;
		T ymax;

		T xmin_ref = T(0.2);
		T ymin_ref = T(0.3);
		T xmax_ref = T(0.3);
		T ymax_ref = T(0.5);

		// Partially overlapped.
		T xmin_test = T(0.1);
		T ymin_test = T(0.1);
		T xmax_test = T(0.3);
		T ymax_test = T(0.4);

		SsdBbox<T>::intersect(xmin_ref, ymin_ref, xmax_ref, ymax_ref, xmin_test, ymin_test, xmax_test, ymax_test, &xmin, &ymin, &xmax, &ymax);
		EXPECT_NEAR(xmin, T(0.2));
		EXPECT_NEAR(ymin, T(0.3));
		EXPECT_NEAR(xmax, T(0.3));
		EXPECT_NEAR(ymax, T(0.4));

		// Fully contain.
		xmin_test = T(0.1);
		ymin_test = T(0.1);
		xmax_test = T(0.4);
		ymax_test = T(0.6);

		SsdBbox<T>::intersect(xmin_ref, ymin_ref, xmax_ref, ymax_ref, xmin_test, ymin_test, xmax_test, ymax_test, &xmin, &ymin, &xmax, &ymax);
		EXPECT_NEAR(xmin, T(0.2));
		EXPECT_NEAR(ymin, T(0.3));
		EXPECT_NEAR(xmax, T(0.3));
		EXPECT_NEAR(ymax, T(0.5));

		// Outside.
		xmin_test = T(0.0);
		ymin_test = T(0.0);
		xmax_test = T(0.1);
		ymax_test = T(0.1);

		SsdBbox<T>::intersect(xmin_ref, ymin_ref, xmax_ref, ymax_ref, xmin_test, ymin_test, xmax_test, ymax_test, &xmin, &ymin, &xmax, &ymax);
		EXPECT_NEAR(xmin, T(0.0));
		EXPECT_NEAR(ymin, T(0.0));
		EXPECT_NEAR(xmax, T(0.0));
		EXPECT_NEAR(ymax, T(0.0));

		return 0;
	}

	long TestBBOX_JaccardOverlap(int nConfig)
	{
		T fxmin1 = T(0.2);
		T fymin1 = T(0.3);
		T fxmax1 = T(0.3);
		T fymax1 = T(0.5);

		// Partially overlapped
		T fxmin2 = T(0.1);
		T fymin2 = T(0.1);
		T fxmax2 = T(0.3);
		T fymax2 = T(0.4);
		T fOverlap = SsdBbox<T>::jaccardOverlap(fxmin1, fymin1, fxmax1, fymax1, fxmin2, fymin2, fxmax2, fymax2);
		T fExpected = T(1.0 / 7);
		EXPECT_NEAR(fOverlap, fExpected);

		// Fully contain
		fxmin2 = T(0.1);
		fymin2 = T(0.1);
		fxmax2 = T(0.4);
		fymax2 = T(0.6);
		fOverlap = SsdBbox<T>::jaccardOverlap(fxmin1, fymin1, fxmax1, fymax1, fxmin2, fymin2, fxmax2, fymax2);
		fExpected = T(2.0 / 15);
		EXPECT_NEAR(fOverlap, fExpected);

		// Outside
		fxmin2 = T(0.0);
		fymin2 = T(0.0);
		fxmax2 = T(0.1);
		fymax2 = T(0.1);
		fOverlap = SsdBbox<T>::jaccardOverlap(fxmin1, fymin1, fxmax1, fymax1, fxmin2, fymin2, fxmax2, fymax2);
		EXPECT_NEAR(fOverlap, T(0));

		return 0;
	}

	long TestBBOX_Match(int nConfig)
	{
		return ERROR_NOT_IMPLEMENTED;
	}

	long TestFindMatches(int nConfig)
	{
		return ERROR_NOT_IMPLEMENTED;
	}

	long TestCountMatches(int nConfig)
	{
		return ERROR_NOT_IMPLEMENTED;
	}

	long TestSoftMax(int nConfig)
	{
		return ERROR_NOT_IMPLEMENTED;
	}

	long TestComputeConfLoss(int nConfig)
	{
		return ERROR_NOT_IMPLEMENTED;
	}

	long TestComputeLocLoss(int nConfig)
	{
		return ERROR_NOT_IMPLEMENTED;
	}

	long TestGetTopKScores(int nConfig)
	{
		return ERROR_NOT_IMPLEMENTED;
	}

	long TestApplyNMS(int nConfig)
	{
		LONG lErr;
		vector<BBOX> bboxes;
		vector<T> scores;
		T fThreshold = T(0);
		int nTopK = 3;
		vector<int> indices;

		if (nConfig > 0)
		{
			return ERROR_NOT_IMPLEMENTED;
		}

		if (lErr = m_ssd.applyNMS(bboxes, scores, fThreshold, nTopK, &indices))
			return lErr;

		return 0;
	}

	long TestMineHardExamples(int nConfig)
	{
		return ERROR_NOT_IMPLEMENTED;
	}
};


//=============================================================================
//	Test Functions
//=============================================================================

template <class T>
long TestSsd<T>::cleanup()
{
	if (m_pObj != NULL)
	{
		delete ((TestData<T>*)m_pObj);
		m_pObj = NULL;
	}

	return 0;
}

template long TestSsd<double>::cleanup();
template long TestSsd<float>::cleanup();


template <class T>
long TestSsd<T>::test_create(int nConfig)
{
	LONG lErr;

	if ((m_pObj = new TestData<T>()) == NULL)
		return ERROR_MEMORY_OUT;

	return ((TestData<T>*)m_pObj)->TestCreate(nConfig);
}

template long TestSsd<double>::test_create(int nConfig);
template long TestSsd<float>::test_create(int nConfig);


//=============================================================================
//	Function Definitions
//=============================================================================

template <class T>
long TestSsd<T>::RunTest(LONG lInput, T* pfInput)
{
	TEST tst = (TEST)(int)pfInput[0];
	int nConfig = 0;

	if (lInput > 1)
		nConfig = (int)pfInput[1];

	try
	{
		LONG lErr;

		if (lErr = test_create(nConfig))
			throw lErr;

		switch (tst)
		{
			case CREATE:
				break;

			case BBOX_SIZE:
				if (lErr = ((TestData<T>*)m_pObj)->TestBBOX_Size(nConfig))
					throw lErr;
				break;

			case BBOX_BOUNDS:
				if (lErr = ((TestData<T>*)m_pObj)->TestBBOX_Bounds(nConfig))
					throw lErr;
				break;

			case BBOX_DIVBOUNDS:
				if (lErr = ((TestData<T>*)m_pObj)->TestBBOX_DivBounds(nConfig))
					throw lErr;
				break;

			case BBOX_CLIP:
				if (lErr = ((TestData<T>*)m_pObj)->TestBBOX_Clip(nConfig))
					throw lErr;
				break;

			case BBOX_DECODE:
				if (lErr = ((TestData<T>*)m_pObj)->TestBBOX_Decode(nConfig))
					throw lErr;
				break;

			case BBOX_ENCODE:
				if (lErr = ((TestData<T>*)m_pObj)->TestBBOX_Encode(nConfig))
					throw lErr;
				break;

			case BBOX_INTERSECT:
				if (lErr = ((TestData<T>*)m_pObj)->TestBBOX_Intersect(nConfig))
					throw lErr;
				break;

			case BBOX_JACCARDOVERLAP:
				if (lErr = ((TestData<T>*)m_pObj)->TestBBOX_JaccardOverlap(nConfig))
					throw lErr;
				break;

			case BBOX_MATCH:
				if (lErr = ((TestData<T>*)m_pObj)->TestBBOX_Match(nConfig))
					throw lErr;
				break;

			case FINDMATCHES:
				if (lErr = ((TestData<T>*)m_pObj)->TestFindMatches(nConfig))
					throw lErr;
				break;

			case COUNTMATCHES:
				if (lErr = ((TestData<T>*)m_pObj)->TestCountMatches(nConfig))
					throw lErr;
				break;

			case SOFTMAX:
				if (lErr = ((TestData<T>*)m_pObj)->TestSoftMax(nConfig))
					throw lErr;
				break;

			case COMPUTE_CONF_LOSS:
				if (lErr = ((TestData<T>*)m_pObj)->TestComputeConfLoss(nConfig))
					throw lErr;
				break;

			case COMPUTE_LOC_LOSS:
				if (lErr = ((TestData<T>*)m_pObj)->TestComputeLocLoss(nConfig))
					throw lErr;
				break;

			case GET_TOPK_SCORES:
				if (lErr = ((TestData<T>*)m_pObj)->TestGetTopKScores(nConfig))
					throw lErr;
				break;

			case APPLYNMS:
				if (lErr = ((TestData<T>*)m_pObj)->TestApplyNMS(nConfig))
					throw lErr;
				break;

			case MINE_HARD_EXAMPLES:
				if (lErr = ((TestData<T>*)m_pObj)->TestMineHardExamples(nConfig))
					throw lErr;
				break;

			default:
				return ERROR_PARAM_OUT_OF_RANGE;
		}

		cleanup();
	}
	catch (long lErrEx)
	{
		cleanup();
		return lErrEx;
	}
	catch (...)
	{
		cleanup();
		return ERROR_SSD;
	}

	return 0;
}

template long TestSsd<double>::RunTest(LONG lInput, double* pfInput);
template long TestSsd<float>::RunTest(LONG lInput, float* pfInput);

// end